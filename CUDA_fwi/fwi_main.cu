#include "fwi_kernel.cuh"
#include "fwi_sched.cuh"

/*
 * In order to generate a source for injection,
 * /system/support/bscgeo/src/wavelet.c
 * functions can be used.
 */
void kernel( propagator_t propagator, real waveletFreq, int shotid, char* outputfolder, char* shotfolder)
{
    double start_t, end_t;
    int stacki;
    real dt,dz,dx,dy;
    integer dimmz, dimmx, dimmy, MaxYPlanesPerWorker, forw_steps, back_steps;

    load_shot_parameters( shotid, &stacki, &dt, &forw_steps, &back_steps,
				&dz, &dx, &dy, 
				&dimmz, &dimmx, &dimmy,
				&MaxYPlanesPerWorker,
				outputfolder, waveletFreq );

    const integer numberOfCells = dimmz * dimmx * dimmy;

    /* set LOCAL integration limits */
    const integer nz0 = 0;
    const integer ny0 = 0;
    const integer nx0 = 0;
    const integer nzf = dimmz;
    const integer nxf = dimmx;
    const integer nyf = dimmy;

    real    *rho;
    v_t     v;
    s_t     s;
    coeff_t coeffs;

    print_debug("The length of local arrays is " I " cells", numberOfCells);

    /* allocate shot memory */
    // alloc_memory_shot  ( numberOfCells, &coeffs, &s, &v, &rho);
    alloc_memory_shot  ( dimmz, dimmx, (nyf - ny0), &coeffs, &s, &v, &rho);

    /* load initial model from a binary file */
    // load_initial_model ( waveletFreq, numberOfCells, &coeffs, &s, &v, rho);
    load_local_velocity_model ( waveletFreq, dimmz, dimmx, ny0, nyf, &coeffs, &s, &v, rho);

    print_debug("dimmx %d,dimmy %d,dimmz %d,numberOfCells %d,numberOfCells * sizeof(real) * WRITTEN_FIELDS %lld",dimmx,dimmy,dimmz,numberOfCells,numberOfCells * sizeof(real) * WRITTEN_FIELDS);
    /* Allocate memory for IO buffer */
    real* io_buffer = (real*) __malloc( ALIGN_REAL, numberOfCells * sizeof(real) * WRITTEN_FIELDS );

    /* inspects every array positions for leaks. Enabled when DEBUG flag is defined */
    // check_memory_shot  ( numberOfCells, &coeffs, &s, &v, rho);
    check_memory_shot  ( dimmz, dimmx, (nyf - ny0), &coeffs, &s, &v, rho);

    switch( propagator )
    {
    case( RTM_KERNEL ):
    {
        start_t = dtime();

        propagate_shot ( FORWARD,
                         v, s, coeffs, rho,
                         forw_steps, back_steps -1,
                         dt,dz,dx,dy,
                         nz0, nzf, nx0, nxf, ny0, nyf,
                         stacki,
                         shotfolder,
                         io_buffer,
                         dimmz, dimmx, dimmy);

        end_t = dtime();

        print_stats("Forward propagation finished in %lf seconds", end_t - start_t );

        start_t = dtime();
        
        propagate_shot ( BACKWARD,
                         v, s, coeffs, rho,
                         forw_steps, back_steps -1,
                         dt,dz,dx,dy,
                         nz0, nzf, nx0, nxf, ny0, nyf,
                         stacki,
                         shotfolder,
                         io_buffer,
                         dimmz, dimmx, dimmy);

        end_t = dtime();

        print_stats("Backward propagation finished in %lf seconds", end_t - start_t );

#ifdef DO_NOT_PERFORM_IO
        print_info("Warning: we are not creating gradient nor preconditioner "
                   "fields, because IO is not enabled for this execution" );
#else
        char fnameGradient[300];
        char fnamePrecond[300];
        sprintf( fnameGradient, "%s/gradient_%05d.dat", shotfolder, shotid );
        sprintf( fnamePrecond , "%s/precond_%05d.dat" , shotfolder, shotid );

        FILE* fgradient = safe_fopen( fnameGradient, (char *)"wb", (char *)__FILE__, __LINE__ );
        FILE* fprecond  = safe_fopen( fnamePrecond , (char *)"wb", (char *)__FILE__, __LINE__ );

        print_info("Storing local preconditioner field in %s", fnameGradient );
        safe_fwrite( io_buffer, sizeof(real), numberOfCells * 12, fgradient, (char *)__FILE__, __LINE__ );

        print_info("Storing local gradient field in %s", fnamePrecond);
        safe_fwrite( io_buffer, sizeof(real), numberOfCells * 12, fprecond , (char *)__FILE__, __LINE__ );

        safe_fclose( fnameGradient, fgradient, (char *)__FILE__, __LINE__ );
        safe_fclose( fnamePrecond , fprecond , (char *)__FILE__, __LINE__ );
#endif

        break;
    }
    case( FM_KERNEL  ):
    {
        start_t = dtime();

        propagate_shot ( FWMODEL,
                         v, s, coeffs, rho,
                         forw_steps, back_steps -1,
                         dt,dz,dx,dy,
                         nz0, nzf, nx0, nxf, ny0, nyf,
                         stacki,
                         shotfolder,
                         io_buffer,
                         dimmz, dimmx, dimmy);

        end_t = dtime();

        print_stats("Forward Modelling finished in %lf seconds", end_t - start_t );
       
        break;
    }
    default:
    {
        print_error("Invalid propagation identifier");
        abort();
    }
    } /* end case */

    // liberamos la memoria alocatada en el shot
    free_memory_shot  ( &coeffs, &s, &v, &rho);
    __free( io_buffer );
};

void gather_shots( char* outputfolder, const real waveletFreq, const int nshots, const int numberOfCells )
{
#ifdef DO_NOT_PERFORM_IO
    print_info("Warning: we are not gathering the results because the IO is disabled "
               "for this execution");
#else
    /* ---------  GLOBAL PRECONDITIONER ACCUMULATION --------- */
    print_info("Gathering local preconditioner fields");

    /* variables for timming */
    double start_t, end_t;

    /* buffers to read and accumulate the fields */
    real* sumbuffer  = (real*)  __malloc( ALIGN_REAL, numberOfCells * sizeof(real) * WRITTEN_FIELDS ); 
    real* readbuffer = (real*)  __malloc( ALIGN_REAL, numberOfCells * sizeof(real) * WRITTEN_FIELDS );
    
    start_t = dtime();

    /* set buffer positions to zero */
    memset ( sumbuffer, 0, numberOfCells * sizeof(real) * WRITTEN_FIELDS );

    for( int shot=0; shot < nshots; shot++)
    {
        char readfilename[300];
        sprintf( readfilename, "%s/shot.%2.1f.%05d/precond_%05d.dat", 
                outputfolder, waveletFreq, shot, shot);

        print_info("Reading preconditioner file '%s'", readfilename );

        FILE* freadfile = safe_fopen( readfilename, (char *)"rb", (char *)__FILE__, __LINE__ );
        safe_fread ( readbuffer, sizeof(real), numberOfCells * WRITTEN_FIELDS, freadfile, (char *)__FILE__, __LINE__ );

        #pragma omp parallel for
#ifdef __INTEL_COMPILER
        #pragma simd
#endif
        for( int i = 0; i < numberOfCells * WRITTEN_FIELDS; i++)
            sumbuffer[i] += readbuffer[i];

        fclose (freadfile);
    }

    char precondfilename[300];
    sprintf( precondfilename, "%s/Preconditioner.%2.1f", outputfolder, waveletFreq );
    FILE* precondfile = safe_fopen( precondfilename, (char *)"wb", (char *)__FILE__, __LINE__ );
    safe_fwrite ( sumbuffer, sizeof(real), numberOfCells * WRITTEN_FIELDS, precondfile, (char *)__FILE__, __LINE__ );
    safe_fclose( precondfilename, precondfile, (char *)__FILE__, __LINE__ );

    end_t = dtime();

    print_stats("Gatering process for preconditioner %s (freq %2.1f) " 
                "completed in: %lf seconds",  
                precondfilename, waveletFreq, end_t - start_t  );

    /* ---------  GLOBAL GRADIENT ACCUMULATION --------- */
    print_info("Gathering local gradient fields");

    start_t = dtime();

    /* set buffer positions to zero */
    memset ( sumbuffer, 0, numberOfCells * sizeof(real) * WRITTEN_FIELDS );

    for( int shot=0; shot < nshots; shot++)
    {
        char readfilename[300];
        sprintf( readfilename, "%s/shot.%2.1f.%05d/gradient_%05d.dat", 
                outputfolder, waveletFreq, shot, shot);

        print_info("Reading gradient file %s", readfilename );

        FILE* freadfile = safe_fopen( readfilename, (char *)"rb", (char *)__FILE__, __LINE__ );
        safe_fread ( readbuffer, sizeof(real), numberOfCells * WRITTEN_FIELDS, freadfile, (char *)__FILE__, __LINE__ );

        #pragma omp parallel for
#ifdef __INTEL_COMPILER
        #pragma simd
#endif
        for( int i = 0; i < numberOfCells * WRITTEN_FIELDS; i++)
            sumbuffer[i] += readbuffer[i];

        fclose (freadfile);
    }

    char gradientfilename[300];
    sprintf( gradientfilename, "%s/Gradient.%2.1f", outputfolder, waveletFreq );
    FILE* gradientfile = safe_fopen( gradientfilename, (char *)"wb", (char *)__FILE__, __LINE__ );
    safe_fwrite ( sumbuffer, sizeof(real), numberOfCells * WRITTEN_FIELDS, gradientfile, (char *)__FILE__, __LINE__ );
    safe_fclose( gradientfilename, gradientfile, (char *)__FILE__, __LINE__ );

    end_t = dtime();

    print_stats("Gatering process for gradient %s (freq %2.1f) "        
                "completed in: %lf seconds", 
                precondfilename, waveletFreq, end_t - start_t  );

    __free(  sumbuffer);
    __free( readbuffer);
#endif
};

int main(int argc, const char* argv[])
{
    double tstart, tend;
    tstart = dtime();

		schedule_t S = load_schedule(argv[1]);

    for(int i=0; i<S.nfreqs; i++) /* frequency loop */
    {
        real waveletFreq   = S.freq[i];
				integer stacki     = S.stacki[i];
				real dt            = S.dt[i];
				integer forw_steps = S.forws[i];
				integer back_steps = S.backs[i];
				real dx            = S.dx[i];
				real dy            = S.dy[i];
				real dz            = S.dz[i];
				integer dimmz      = S.dimmz[i];
				integer dimmy      = S.dimmy[i];
				integer dimmx      = S.dimmx[i];
//				integer ppd        = S.ppd[i];
//				integer nworkers   = S.nworkers[i];
				integer MaxYPlanesPerWorker = S.ppd[i];

        print_info("\n------ Computing %d-th frequency (%.2fHz).  -----\n", i, waveletFreq); 
				
        const integer numberOfCells = dimmz * dimmx * dimmx;
        const size_t VolumeMemory  = numberOfCells * sizeof(real) * 58;

        print_stats("Local domain size for freq %f [%d][%d][%d] is %lu bytes (%lf GB)", 
                    waveletFreq, dimmz, dimmx, dimmy, VolumeMemory, TOGB(VolumeMemory) );

        for(int grad=0; grad<S.ngrads; grad++) /* iteracion de inversion */
        {
            print_info("Processing %d-gradient iteration", grad);

            for(int shot=0; shot<S.nshots; shot++)
            {
                char shotfolder[200];
                sprintf(shotfolder, "%s/shot.%2.1f.%05d", S.outputfolder, waveletFreq, shot);
                create_folder( shotfolder );

                store_shot_parameters ( shot, &stacki, &dt, &forw_steps, &back_steps, 
                                        &dz, &dx, &dy, 
                                        &dimmz, &dimmx, &dimmy, 
																				&MaxYPlanesPerWorker,
                                        S.outputfolder, waveletFreq );

                kernel( RTM_KERNEL, waveletFreq, shot, S.outputfolder, shotfolder);

                print_info("\tGradient loop processed for %d-th shot", shot);
                
                //update_shot()
            }

            gather_shots( S.outputfolder, waveletFreq, S.nshots, numberOfCells );

            for(int test=0; test<S.ntests; test++)
            {
                print_info("\tProcessing %d-th test iteration", test);
                
                for(int shot=0; shot<S.nshots; shot++)
                {
                    char shotfolder[200];
                    sprintf(shotfolder, "%s/test.%05d.shot.%2.1f.%05d", 
                            S.outputfolder, test, waveletFreq, shot);
                    create_folder( shotfolder );
                    
                    store_shot_parameters ( shot, &stacki, &dt, &forw_steps, &back_steps, 
                                            &dz, &dx, &dy, 
                                            &dimmz, &dimmx, &dimmy,
																					 	&MaxYPlanesPerWorker,	
                                            S.outputfolder, waveletFreq );

                    kernel( FM_KERNEL , waveletFreq, shot, S.outputfolder, shotfolder);
                
                    print_info("\t\tTest loop processed for the %d-th shot", shot);
                }
            } /* end of test loop */
        } /* end of gradient loop */
    } /* end of frequency loop */

    tend = dtime() - tstart;

		print_info("FWI Program finished in %lf seconds\n", tend);

    return 0;
}
