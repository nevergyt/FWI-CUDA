#include "hip/hip_runtime.h"
#include "fwi_common.cuh"

int gettimeofday(struct timeval *tv, struct timezone *tz)
{
    FILETIME ft;
    GetSystemTimeAsFileTime(&ft);

    ULARGE_INTEGER uliTime;
    uliTime.LowPart = ft.dwLowDateTime;
    uliTime.HighPart = ft.dwHighDateTime;
    uint64_t ullTime = uliTime.QuadPart;

    static const uint64_t EPOCH_DIFFERENCE = 116444736000000000ULL;
    ullTime -= EPOCH_DIFFERENCE;
    ullTime /= 10;

    tv->tv_sec = ullTime / 1000000;
    tv->tv_usec = ullTime % 1000000;

    return 0;
}

int max_int( int a, int b)
{
    return ((a >= b) ? a : b);
};

inline double dtime(void)
{
    double tseconds = 0.0;
    struct timeval mytime;
    gettimeofday( &mytime, (struct timezone*) 0);
    tseconds = (double) (mytime.tv_sec + (double) mytime.tv_usec * 1.0e-6);
    return (tseconds);
};

double TOGB(size_t bytes)
{
    return (bytes / (1024.f * 1024.f * 1024.f));
};

void read_fwi_parameters (const char *fname,
                          real *lenz,
                          real *lenx,
                          real *leny,
                          real *vmin,
                          real *srclen,
                          real *rcvlen,
													int  *nshots,
													int  *ngrads,
													int  *ntests,
													real  *workmem,
													real  *slavemem,
                          char *outputfolder)
{
    FILE *fp = safe_fopen(fname, (char *)"r", (char *)__FILE__, __LINE__ );

    CHECK( fscanf( fp, "%f\n", (real*) lenz   ) );
    CHECK( fscanf( fp, "%f\n", (real*) lenx   ) );
    CHECK( fscanf( fp, "%f\n", (real*) leny   ) );
    CHECK( fscanf( fp, "%f\n", (real*) vmin   ) );
    CHECK( fscanf( fp, "%f\n", (real*) srclen ) );
    CHECK( fscanf( fp, "%f\n", (real*) rcvlen ) );
    CHECK( fscanf( fp, "%d\n", (int*)  nshots  ) );
    CHECK( fscanf( fp, "%d\n", (int*)  ngrads  ) );
    CHECK( fscanf( fp, "%d\n", (int*)  ntests  ) );
    CHECK( fscanf( fp, "%f\n", (real*)  workmem ) );
    CHECK( fscanf( fp, "%f\n", (real*)  slavemem) );
    CHECK( fscanf( fp, "%s\n",  outputfolder  ) );

		print_debug("Worker memory %f slave memory %f", *workmem, *slavemem);

    print_debug("Len (z,x,y) (%.2f,%.2f,%.2f)\n \
				vmin %.2f scrlen %.2f rcvlen %.2f outputfolder '%s'\n \
				worker memory %.5f GB slave memory %.5fGB",
      *lenz, *lenx, *leny, *vmin, *srclen, *rcvlen, outputfolder, *workmem, *slavemem );

    fclose(fp);
};

/*
  This function is intended to round up a number (number) to the nearest multiple of the register
  size. In this way, we assure that the dimensions of the domain are suited to the most aggressive
  compiler optimizations.
 */
integer roundup(integer number, integer multiple)
{
    if (multiple == 0)
        return number;

    int remainder = number % multiple;
    if (remainder == 0)
        return number;

    return number + multiple - remainder;
};

/*
 NAME:allocate_shot_memory
 PURPOSE: Create files to store final preconditioner and gradient results. Must be initialized with zeroes.

 outputfolder     (in) folder where snapshot data is store
 VolumeMemory     (in) memory needed to store the domain

 RETURN none
 */
void create_output_volumes(char *outputfolder, integer VolumeMemory)
{
    print_debug("Creating output files in %s", outputfolder);

#ifdef DO_NOT_PERFORM_IO
    print_info("Warning: we are not doing any IO here.");
#else
    char fnamePrecond[300], fnameGradient[300];

    sprintf( fnameGradient, "%s/resultGradient.res", outputfolder);
    sprintf( fnamePrecond , "%s/resultPrecond.res", outputfolder);

    FILE *fGradient = safe_fopen( fnameGradient, (char *)"wb", (char *)__FILE__, __LINE__ );
    FILE *fPrecond  = safe_fopen( fnamePrecond , (char *)"wb", (char *)__FILE__, __LINE__ );

    int numIts = ceil( VolumeMemory / IO_CHUNK_SIZE );

    /* create buffer array */
    real *tmparray = (real*) __malloc( ALIGN_REAL, IO_CHUNK_SIZE );

    /* perform the accumulation of the chunks */
    for (int i=0; i<numIts; i++) {
        safe_fwrite(tmparray, 1, IO_CHUNK_SIZE, fGradient, (char *)__FILE__, __LINE__ );
        safe_fwrite(tmparray, 1, IO_CHUNK_SIZE, fPrecond , (char *)__FILE__, __LINE__ );
    }

    __free(tmparray);

    // close files
    safe_fclose( fnameGradient, fGradient, (char *)__FILE__, __LINE__ );
    safe_fclose( fnamePrecond , fPrecond , (char *)__FILE__, __LINE__ );
#endif
}

/*
 NAME:create_folder
 PURPOSE:During execution creates temporal folders to organize necessary data for the execution

 folder      (in) name of the temporal folder created
 parent_rank (in) name of the rank related to the data archived in to the folder
 shotID      (in) identifier of the shot related to the data to be archived in to the folder

 RETURN none
 */
void create_folder(const char *folder)
{
    print_debug("Creating '%s'",folder);
    
		if (mkdir_p(folder) != 0) {
        print_error("cant create folder %s (Err code: %s)", folder, strerror(errno));
        exit(-1);
    }
    print_debug("Folder '%s' created",folder);
};

/*
 NAME: mkdir_p
 PURPOSE: creates the hierarchy of folders requested, if they do not exist.

 RETURN 0 if successful, !=0 otherwise
 */
int mkdir_p(const char *dir)
{
    char tmp[256];
    char *p = NULL;
    size_t len;

    snprintf(tmp, sizeof(tmp),"%s",dir);
    len = strlen(tmp);

    if(tmp[len - 1] == '/')
        tmp[len - 1] = 0;

    for(p = tmp + 1; *p; p++) {
        if(*p == '/') {
            *p = 0;
            int rc = mkdir(tmp);
            if (rc != 0 && errno != EEXIST) {
                print_error("Error creating folder %s (Err code %s)", tmp, strerror(errno));
                return -1;
            }

            *p = '/';
        }
    }
    int rc = mkdir(tmp);
    if (rc != 0 && errno != EEXIST) {
        print_error("Error creating folder %s (Err code %s)", tmp, strerror(errno));
        return -1;
    }

    return 0;
}

void store_shot_parameters(int     shotid,
                           int     *stacki,
                           real    *dt,
                           int     *nt_fwd,
                           int     *nt_bwd,
                           real    *dz,
                           real    *dx,
                           real    *dy,
                           integer *dimmz,
                           integer *dimmx,
                           integer *dimmy,
													 integer *LocalYPlanes,
                           char    *outputfolder, 
                           real    waveletFreq)
{
    char name[200];

    sprintf(name, "%s/shotparams_%2.1f.%05d.dat", 
            outputfolder, waveletFreq, shotid);

    print_debug("Storing parameters for freq %.3fHz, shot %d into %s", waveletFreq, shotid, name);

    FILE *fp = safe_fopen(name, (char *)"w", (char *)__FILE__, __LINE__);

    fprintf(fp, "%f\n",  (real   ) *dz     );
    fprintf(fp, "%f\n",  (real   ) *dx     );
    fprintf(fp, "%f\n",  (real   ) *dy     );
    fprintf(fp,  I"\n", (integer) *dimmz  );
    fprintf(fp,  I"\n", (integer) *dimmx  );
    fprintf(fp,  I"\n", (integer) *dimmy  );
    fprintf(fp,  I"\n", (integer) *LocalYPlanes  );
    fprintf(fp, "%d\n",  (int    ) *nt_fwd );
    fprintf(fp, "%d\n",  (int    ) *nt_bwd );
    fprintf(fp, "%f\n",  (real   ) *dt     );
    fprintf(fp, "%d\n",  (int    ) *stacki );
		
		print_debug("Local Y planes %d", *LocalYPlanes);

    safe_fclose(name, fp, (char *)__FILE__, __LINE__ );
};

void load_shot_parameters(int     shotid,
                          int     *stacki,
                          real    *dt,
                          int     *nt_fwd,
                          int     *nt_bwd,
                          real    *dz,
                          real    *dx,
                          real    *dy,
                          integer *dimmz,
                          integer *dimmx,
                          integer *dimmy,
                          integer *LocalYPlanes,
                          char    *outputfolder,
                          real    waveletFreq)
{
    char name[200];

    sprintf(name, "%s/shotparams_%2.1f.%05d.dat", outputfolder, waveletFreq, shotid);
    print_debug("Loading parameters for freq %.3fHz, shot %d into %s", waveletFreq, shotid, name);

    FILE *fp = safe_fopen(name, (char *)"r", (char *)__FILE__, __LINE__);

    CHECK( fscanf(fp, "%f\n",  (real*   ) dz     ) );
    CHECK( fscanf(fp, "%f\n",  (real*   ) dx     ) );
    CHECK( fscanf(fp, "%f\n",  (real*   ) dy     ) );
    CHECK( fscanf(fp,  I"\n",  (integer*) dimmz  ) );
    CHECK( fscanf(fp,  I"\n",  (integer*) dimmx  ) );
    CHECK( fscanf(fp,  I"\n",  (integer*) dimmy  ) );
    CHECK( fscanf(fp,  I"\n",  (integer*) LocalYPlanes  ) );
    CHECK( fscanf(fp, "%d\n",  (int*    ) nt_fwd ) );
    CHECK( fscanf(fp, "%d\n",  (int*    ) nt_bwd ) );
    CHECK( fscanf(fp, "%f\n",  (real*   ) dt     ) );
    CHECK( fscanf(fp, "%d\n",  (int*    ) stacki ) );

		print_debug("Local Y planes %d,dimmmx %d,dimmz %d,dimmy %d", *LocalYPlanes,*dimmx,*dimmz,*dimmy);

    safe_fclose( name, fp, (char *)__FILE__, __LINE__);
};

void load_freqlist( const char* filename, int *nfreqs, real **freqlist )
{
    int count  = 0;
    real freq;

    FILE *freqfile = safe_fopen( filename, (char *)"r", (char *)__FILE__, __LINE__);

    while( 1 )
    {
        int n = fscanf( freqfile, "%f", &freq);

        if ( n == 1 )
        {
            count += 1;
        }
        else if (errno != 0)
        {
            print_error("Error while reading freqlist file");
            break;
        }
        else if ( n == EOF )
        {
            break;
        }
    }


    /* Allocate memory for frequencies */
    *freqlist = (real*) __malloc( ALIGN_REAL, count * sizeof(real));

    /* return to initial position */
    fseek( freqfile, 0, SEEK_SET);
    count = 0;



    /* read again the file, storing the wavelet frequencies this time */
    while( 1 )
    {
        int n = fscanf( freqfile, "%f", &freq);

        if ( n == 1 )
        {
            (*freqlist)[count++] = freq;
        }
        else if (errno != 0)
        {
            print_error("Error while reading freqlist file");
            break;
        }
        else if ( n == EOF )
        {
            break;
        }
    }
    fclose( freqfile );

    *nfreqs = count;

    print_info("A total of %d frequencies were found...", *nfreqs );
    for( int i=0; i<count; i++)
        print_info("     %.2f Hz", (*freqlist)[i] );
};

void* __malloc( size_t alignment, const integer size)
{
    print_debug("alignment %d ,size %d",alignment,size);
    void *buffer;
    int error;

    if( (error=posix_memalign( &buffer, alignment, size)) != 0)
    {
        print_error("Cant allocate buffer correctly");
        abort();
    }


    return (buffer);
};

void __free ( void* ptr)
{
#if defined(__INTEL_COMPILER)
	_mm_free(ptr);
#else
    _aligned_free( ptr );
#endif
};

/*
	Reads an environmental variable.
 */
char *read_env_variable (const char* varname)
{	
	char* s = getenv(varname);
	
	if ( s == NULL )
	{
		fprintf(stderr, "%s: ERROR: unable to read  %s env. var\n", __FUNCTION__, varname);
		abort();
	}

#ifdef DEBUG
	printf("%s: %s variable value is :%s\n", __FUNCTION__, varname, s);
#endif

	return (s);
};


FILE* safe_fopen(const char *filename, char *mode, char* srcfilename, int linenumber)
{
    FILE* temp = fopen( filename, mode);
    
    if( temp == NULL){
        print_error("Cant open filename %s, openmode '%s' (called from %s - %d)", filename, mode, srcfilename, linenumber);
        perror("cant open");
        exit(-1);
    }
    return temp;
};

void safe_fclose ( const char *filename, FILE* stream, char* srcfilename, int linenumber)
{
    if ( fclose( stream ) != 0)
    {
        print_error("Cant close filename %s (called from %s - %d)", filename, srcfilename, linenumber);
        abort();
    }

/*if ( unlink(filename)  != 0)
  {
    fprintf(stderr, "%s:%d: Cant unlink file %s correctly!\n", srcfilename, linenumber, filename );
    abort();
  }*/
};


inline void safe_fwrite (void *ptr, size_t size, size_t nmemb, FILE *stream, char* srcfilename, int linenumber)
{
#if defined(DO_NOT_PERFORM_IO)
    print_info("Warning: we are not doing any IO (called from %s).", __FUNCTION__);
#else
    if( stream == NULL ){
        print_error("Invalid stream\n");
        abort();
    }
    double start_t = dtime();
    size_t res = fwrite( ptr, size, nmemb, stream);
    double end_t = dtime() - start_t;
    
   
    if( res != nmemb )
    {
        print_error("Error while fwrite (called from %s - %d)", srcfilename, linenumber );
        abort();
    }
    
    double mbytes = (1.0 * size * nmemb) / (1024.0 * 1024.0);
		print_stats("WRITE Time %lf, elements %lu bytes %lu, MB %lf MB/s %lf", end_t, nmemb, size*nmemb, mbytes, mbytes / end_t);
#endif
};

void safe_fread (void *ptr, size_t size, size_t nmemb, FILE *stream, char* srcfilename, int linenumber)
{

    if( stream == NULL ){
        print_error("Invalid\n");
        abort();
    }
    
		double start_t = dtime();
		size_t res = fread( ptr, size, nmemb, stream);
		double end_t = dtime() - start_t;

    
		if( res != nmemb )
    {
        print_error("Error while fread (called from %s - %d)", srcfilename, linenumber);
        print_error("Trying to read %lu elements, only %lu were recovered", nmemb, res);
        abort();
    }
    
    double mbytes = (1.0 * size * nmemb) / (1024.0 * 1024.0);
		print_stats("READ Time %lf, elements %lu bytes %lu, MB %lf MB/s %lf", end_t, nmemb, size*nmemb, mbytes, mbytes / end_t );

};



void fwi_writelog(const char *SourceFileName, 
                  const int LineNumber,
                  const char *FunctionName,
                  const char* MessageHeader,
                  const char *fmt,
                  ...)
{
    const char LogFileName[] = "./fwi.log";
    
    FILE *fp = safe_fopen ( LogFileName, (char *)"a", (char *)__FILE__, __LINE__ );
    
    va_list args;
    va_start(args, fmt);
    fprintf(fp, "%s :[%s:%d:%s] :: ", MessageHeader, SourceFileName, LineNumber, FunctionName );
    vfprintf(fp, fmt, args);
    fprintf(fp, "\n");
    va_end(args);
    
    safe_fclose ( LogFileName, fp, (char *)__FILE__, __LINE__);
};
