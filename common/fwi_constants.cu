#include "fwi_constants.cuh"

/* extern variables declared in the header file */
const integer  WRITTEN_FIELDS =   12; /* >= 12.  */
const integer  HALO           =    4; /* >= 4    */ 
const integer  SIMD_LENGTH    =    8; /* # of real elements fitting into regs */
const real     IT_FACTOR      = 0.02;
const real     IO_CHUNK_SIZE  = 1024.f * 1024.f;

const size_t ALIGN_INT     = 16;
const size_t ALIGN_INTEGER = 16;
const size_t ALIGN_REAL    = 64;


