#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  fwi_kernel.c
 *
 *    Description:  kernel propagator implementation
 *
 *        Version:  1.0
 *        Created:  14/12/15 12:10:05
 *       Revision:  none
 *       Compiler:  icc
 *
 *         Author:  YOUR NAME (),
 *   Organization:
 *
 * =====================================================================================
 */

#include "fwi_kernel.cuh"


/*
 * Initializes an array of length "length" to a random number.
 */
//__global__
void
set_array_to_random_real( real* __restrict__ array, const integer length)
{
    const real randvalue = rand() / (1.0 * RAND_MAX);

    //todo:
    int i = blockDim.x *
            blockIdx.x +
            threadIdx.x;

    if (i < length)
    {
        array[i]= randvalue;
    }

}

/*
 * Initializes an array of length "length" to a constant floating point value.
 */
//__global__
void set_array_to_constant( real* __restrict__ array, const real value, const integer length)
{
    int i = blockDim.x *
            blockIdx.x +
            threadIdx.x;

    if (i < length)
    {
        array[i]= value;
    }
}

void check_memory_shot( const integer dimmz,
												const integer dimmx,
												const integer dimmy,
                        coeff_t *c,
                        s_t     *s,
                        v_t     *v,
                        real    *rho)
{
#if defined(DEBUG)
    print_debug("Checking memory shot values");

    real UNUSED(value);
    for( int i=0; i < (dimmz * dimmx * dimmy); i++)
    {
        value = c->c11[i];
        value = c->c12[i];
        value = c->c13[i];
        value = c->c14[i];
        value = c->c15[i];
        value = c->c16[i];

        value = c->c22[i];
        value = c->c23[i];
        value = c->c24[i];
        value = c->c25[i];
        value = c->c26[i];

        value = c->c33[i];
        value = c->c34[i];
        value = c->c35[i];
        value = c->c36[i];

        value = c->c44[i];
        value = c->c45[i];
        value = c->c46[i];
        
        value = c->c55[i];
        value = c->c56[i];
        value = c->c66[i];

        value = v->tl.u[i];
        value = v->tl.v[i];
        value = v->tl.w[i];

        value = v->tr.u[i];
        value = v->tr.v[i];
        value = v->tr.w[i];

        value = v->bl.u[i];
        value = v->bl.v[i];
        value = v->bl.w[i];
        
        value = v->br.u[i];
        value = v->br.v[i];
        value = v->br.w[i];

        value = rho[i];
    }
		print_debug("Shot memory is well allocated");
#endif
};

void alloc_memory_shot( const integer dimmz,
												const integer dimmx,
												const integer dimmy,
                        coeff_t *c,
                        s_t     *s,
                        v_t     *v,
                        real    **rho)
{
    const size_t size = dimmz * dimmx * dimmy * sizeof(real);

    print_debug("ptr size = %lu bytes (%lu elements)", 
				size, 
				(size_t) dimmz * dimmx * dimmy );

    /* allocate coefficients */
    c->c11 = (real*) __malloc( ALIGN_REAL, size);
    c->c12 = (real*) __malloc( ALIGN_REAL, size);
    c->c13 = (real*) __malloc( ALIGN_REAL, size);
    c->c14 = (real*) __malloc( ALIGN_REAL, size);
    c->c15 = (real*) __malloc( ALIGN_REAL, size);
    c->c16 = (real*) __malloc( ALIGN_REAL, size);

    c->c22 = (real*) __malloc( ALIGN_REAL, size);
    c->c23 = (real*) __malloc( ALIGN_REAL, size);
    c->c24 = (real*) __malloc( ALIGN_REAL, size);
    c->c25 = (real*) __malloc( ALIGN_REAL, size);
    c->c26 = (real*) __malloc( ALIGN_REAL, size);

    c->c33 = (real*) __malloc( ALIGN_REAL, size);
    c->c34 = (real*) __malloc( ALIGN_REAL, size);
    c->c35 = (real*) __malloc( ALIGN_REAL, size);
    c->c36 = (real*) __malloc( ALIGN_REAL, size);

    c->c44 = (real*) __malloc( ALIGN_REAL, size);
    c->c45 = (real*) __malloc( ALIGN_REAL, size);
    c->c46 = (real*) __malloc( ALIGN_REAL, size);

    c->c55 = (real*) __malloc( ALIGN_REAL, size);
    c->c56 = (real*) __malloc( ALIGN_REAL, size);
    c->c66 = (real*) __malloc( ALIGN_REAL, size);

    /* allocate velocity components */
    v->tl.u = (real*) __malloc( ALIGN_REAL, size);
    v->tl.v = (real*) __malloc( ALIGN_REAL, size);
    v->tl.w = (real*) __malloc( ALIGN_REAL, size);

    v->tr.u = (real*) __malloc( ALIGN_REAL, size);
    v->tr.v = (real*) __malloc( ALIGN_REAL, size);
    v->tr.w = (real*) __malloc( ALIGN_REAL, size);

    v->bl.u = (real*) __malloc( ALIGN_REAL, size);
    v->bl.v = (real*) __malloc( ALIGN_REAL, size);
    v->bl.w = (real*) __malloc( ALIGN_REAL, size);

    v->br.u = (real*) __malloc( ALIGN_REAL, size);
    v->br.v = (real*) __malloc( ALIGN_REAL, size);
    v->br.w = (real*) __malloc( ALIGN_REAL, size);

    /* allocate stress components   */
    s->tl.zz = (real*) __malloc( ALIGN_REAL, size);
    s->tl.xz = (real*) __malloc( ALIGN_REAL, size);
    s->tl.yz = (real*) __malloc( ALIGN_REAL, size);
    s->tl.xx = (real*) __malloc( ALIGN_REAL, size);
    s->tl.xy = (real*) __malloc( ALIGN_REAL, size);
    s->tl.yy = (real*) __malloc( ALIGN_REAL, size);

    s->tr.zz = (real*) __malloc( ALIGN_REAL, size);
    s->tr.xz = (real*) __malloc( ALIGN_REAL, size);
    s->tr.yz = (real*) __malloc( ALIGN_REAL, size);
    s->tr.xx = (real*) __malloc( ALIGN_REAL, size);
    s->tr.xy = (real*) __malloc( ALIGN_REAL, size);
    s->tr.yy = (real*) __malloc( ALIGN_REAL, size);

    s->bl.zz = (real*) __malloc( ALIGN_REAL, size);
    s->bl.xz = (real*) __malloc( ALIGN_REAL, size);
    s->bl.yz = (real*) __malloc( ALIGN_REAL, size);
    s->bl.xx = (real*) __malloc( ALIGN_REAL, size);
    s->bl.xy = (real*) __malloc( ALIGN_REAL, size);
    s->bl.yy = (real*) __malloc( ALIGN_REAL, size);

    s->br.zz = (real*) __malloc( ALIGN_REAL, size);
    s->br.xz = (real*) __malloc( ALIGN_REAL, size);
    s->br.yz = (real*) __malloc( ALIGN_REAL, size);
    s->br.xx = (real*) __malloc( ALIGN_REAL, size);
    s->br.xy = (real*) __malloc( ALIGN_REAL, size);
    s->br.yy = (real*) __malloc( ALIGN_REAL, size);

    /* allocate density array       */
    *rho = (real*) __malloc( ALIGN_REAL, size);
};

void free_memory_shot( coeff_t *c,
                       s_t     *s,
                       v_t     *v,
                       real    **rho)
{
    /* deallocate coefficients */
    __free( (void*) c->c11 );
    __free( (void*) c->c12 );
    __free( (void*) c->c13 );
    __free( (void*) c->c14 );
    __free( (void*) c->c15 );
    __free( (void*) c->c16 );

    __free( (void*) c->c22 );
    __free( (void*) c->c23 );
    __free( (void*) c->c24 );
    __free( (void*) c->c25 );
    __free( (void*) c->c26 );
    __free( (void*) c->c33 );

    __free( (void*) c->c34 );
    __free( (void*) c->c35 );
    __free( (void*) c->c36 );

    __free( (void*) c->c44 );
    __free( (void*) c->c45 );
    __free( (void*) c->c46 );

    __free( (void*) c->c55 );
    __free( (void*) c->c56 );

    __free( (void*) c->c66 );

    /* deallocate velocity components */
    __free( (void*) v->tl.u );
    __free( (void*) v->tl.v );
    __free( (void*) v->tl.w );

    __free( (void*) v->tr.u );
    __free( (void*) v->tr.v );
    __free( (void*) v->tr.w );

    __free( (void*) v->bl.u );
    __free( (void*) v->bl.v );
    __free( (void*) v->bl.w );

    __free( (void*) v->br.u );
    __free( (void*) v->br.v );
    __free( (void*) v->br.w );

    /* deallocate stres components   */
    __free( (void*) s->tl.zz );
    __free( (void*) s->tl.xz );
    __free( (void*) s->tl.yz );
    __free( (void*) s->tl.xx );
    __free( (void*) s->tl.xy );
    __free( (void*) s->tl.yy );

    __free( (void*) s->tr.zz );
    __free( (void*) s->tr.xz );
    __free( (void*) s->tr.yz );
    __free( (void*) s->tr.xx );
    __free( (void*) s->tr.xy );
    __free( (void*) s->tr.yy );

    __free( (void*) s->bl.zz );
    __free( (void*) s->bl.xz );
    __free( (void*) s->bl.yz );
    __free( (void*) s->bl.xx );
    __free( (void*) s->bl.xy );
    __free( (void*) s->bl.yy );

    __free( (void*) s->br.zz );
    __free( (void*) s->br.xz );
    __free( (void*) s->br.yz );
    __free( (void*) s->br.xx );
    __free( (void*) s->br.xy );
    __free( (void*) s->br.yy );


    /* deallocate density array       */
    __free( (void*) *rho );
};

/*
 * Loads initial values from coeffs, stress and velocity.
 *
 * dimmz: number of z planes.
 * dimmx: number of x planes
 * FirstYPlane: first Y plane of my local domain (includes HALO)
 * LastYPlane: last Y plane of my local domain (includes HALO)
 */
void load_local_velocity_model ( const real    waveletFreq,
													const integer dimmz,
													const integer dimmx,
													const integer FirstYPlane,
													const integer LastYPlane,
                          coeff_t *c,
                          s_t     *s,
                          v_t     *v,
                          real    *rho)
{
	/* Local variables */
	  double tstart_outer, tstart_inner, tend_outer, tend_inner;
    double iospeed_inner, iospeed_outer;
    char modelname[300];

    const integer cellsInVolume  = dimmz * dimmx * (LastYPlane - FirstYPlane);  
    const integer bytesForVolume = WRITTEN_FIELDS * cellsInVolume * sizeof(real);
    
		/*
		 * Material, velocities and stresses are initizalized
		 * accorting to the compilation flags, either randomly
		 * or by reading an input velocity model.
		 */

		/* initialize stress arrays */
    set_array_to_constant( s->tl.zz, 0, cellsInVolume);
    set_array_to_constant( s->tl.xz, 0, cellsInVolume);
    set_array_to_constant( s->tl.yz, 0, cellsInVolume);
    set_array_to_constant( s->tl.xx, 0, cellsInVolume);
    set_array_to_constant( s->tl.xy, 0, cellsInVolume);
    set_array_to_constant( s->tl.yy, 0, cellsInVolume);
    set_array_to_constant( s->tr.zz, 0, cellsInVolume);
    set_array_to_constant( s->tr.xz, 0, cellsInVolume);
    set_array_to_constant( s->tr.yz, 0, cellsInVolume);
    set_array_to_constant( s->tr.xx, 0, cellsInVolume);
    set_array_to_constant( s->tr.xy, 0, cellsInVolume);
    set_array_to_constant( s->tr.yy, 0, cellsInVolume);
    set_array_to_constant( s->bl.zz, 0, cellsInVolume);
    set_array_to_constant( s->bl.xz, 0, cellsInVolume);
    set_array_to_constant( s->bl.yz, 0, cellsInVolume);
    set_array_to_constant( s->bl.xx, 0, cellsInVolume);
    set_array_to_constant( s->bl.xy, 0, cellsInVolume);
    set_array_to_constant( s->bl.yy, 0, cellsInVolume);
    set_array_to_constant( s->br.zz, 0, cellsInVolume);
    set_array_to_constant( s->br.xz, 0, cellsInVolume);
    set_array_to_constant( s->br.yz, 0, cellsInVolume);
    set_array_to_constant( s->br.xx, 0, cellsInVolume);
    set_array_to_constant( s->br.xy, 0, cellsInVolume);
    set_array_to_constant( s->br.yy, 0, cellsInVolume);

#if defined(DO_NOT_PERFORM_IO)

    /* initialize material coefficients */
    set_array_to_random_real( c->c11, cellsInVolume);
    set_array_to_random_real( c->c12, cellsInVolume);
    set_array_to_random_real( c->c13, cellsInVolume);
    set_array_to_random_real( c->c14, cellsInVolume);
    set_array_to_random_real( c->c15, cellsInVolume);
    set_array_to_random_real( c->c16, cellsInVolume);
    set_array_to_random_real( c->c22, cellsInVolume);
    set_array_to_random_real( c->c23, cellsInVolume);
    set_array_to_random_real( c->c24, cellsInVolume);
    set_array_to_random_real( c->c25, cellsInVolume);
    set_array_to_random_real( c->c26, cellsInVolume);
    set_array_to_random_real( c->c33, cellsInVolume);
    set_array_to_random_real( c->c34, cellsInVolume);
    set_array_to_random_real( c->c35, cellsInVolume);
    set_array_to_random_real( c->c36, cellsInVolume);
    set_array_to_random_real( c->c44, cellsInVolume);
    set_array_to_random_real( c->c45, cellsInVolume);
    set_array_to_random_real( c->c46, cellsInVolume);
    set_array_to_random_real( c->c55, cellsInVolume);
    set_array_to_random_real( c->c56, cellsInVolume);
    set_array_to_random_real( c->c66, cellsInVolume);
    
    /* initalize velocity components */
    set_array_to_random_real( v->tl.u, cellsInVolume );
    set_array_to_random_real( v->tl.v, cellsInVolume );
    set_array_to_random_real( v->tl.w, cellsInVolume );
    set_array_to_random_real( v->tr.u, cellsInVolume );
    set_array_to_random_real( v->tr.v, cellsInVolume );
    set_array_to_random_real( v->tr.w, cellsInVolume );
    set_array_to_random_real( v->bl.u, cellsInVolume );
    set_array_to_random_real( v->bl.v, cellsInVolume );
    set_array_to_random_real( v->bl.w, cellsInVolume );
    set_array_to_random_real( v->br.u, cellsInVolume );
    set_array_to_random_real( v->br.v, cellsInVolume );
    set_array_to_random_real( v->br.w, cellsInVolume );

    /* initialize density (rho) */
    set_array_to_random_real( rho, cellsInVolume );

#else /* load velocity model from external file */
    
    /* initialize material coefficients */
    set_array_to_constant( c->c11, 1.0, cellsInVolume);
    set_array_to_constant( c->c12, 1.0, cellsInVolume);
    set_array_to_constant( c->c13, 1.0, cellsInVolume);
    set_array_to_constant( c->c14, 1.0, cellsInVolume);
    set_array_to_constant( c->c15, 1.0, cellsInVolume);
    set_array_to_constant( c->c16, 1.0, cellsInVolume);
    set_array_to_constant( c->c22, 1.0, cellsInVolume);
    set_array_to_constant( c->c23, 1.0, cellsInVolume);
    set_array_to_constant( c->c24, 1.0, cellsInVolume);
    set_array_to_constant( c->c25, 1.0, cellsInVolume);
    set_array_to_constant( c->c26, 1.0, cellsInVolume);
    set_array_to_constant( c->c33, 1.0, cellsInVolume);
    set_array_to_constant( c->c34, 1.0, cellsInVolume);
    set_array_to_constant( c->c35, 1.0, cellsInVolume);
    set_array_to_constant( c->c36, 1.0, cellsInVolume);
    set_array_to_constant( c->c44, 1.0, cellsInVolume);
    set_array_to_constant( c->c45, 1.0, cellsInVolume);
    set_array_to_constant( c->c46, 1.0, cellsInVolume);
    set_array_to_constant( c->c55, 1.0, cellsInVolume);
    set_array_to_constant( c->c56, 1.0, cellsInVolume);
    set_array_to_constant( c->c66, 1.0, cellsInVolume);

    /* initialize density (rho) */
    set_array_to_constant( rho, 1.0, cellsInVolume );

    char* fwipath = read_env_variable("FWIDIR");
     /* open initial model, binary file */
    sprintf( modelname, "%s/InputModels/velocitymodel_(%.2f).bin",fwipath, waveletFreq );
    print_info("Loading input model %s from disk (this could take a while)", modelname);

    /* start clock, take into account file opening */
    tstart_outer = dtime();
    FILE* model = safe_fopen( modelname, (char *)"rb", (char *)__FILE__, __LINE__ );

    /* seek to the correct position corresponding to mpi_rank */
    fseek ( model, sizeof(real) * WRITTEN_FIELDS * dimmz * dimmx * FirstYPlane, SEEK_SET);
    
    /* start clock, do not take into account file opening */
    tstart_inner = dtime();

    /* initalize velocity components */
    safe_fread( v->tl.u, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->tl.v, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->tl.w, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->tr.u, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->tr.v, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->tr.w, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->bl.u, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->bl.v, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->bl.w, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->br.u, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->br.v, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );
    safe_fread( v->br.w, sizeof(real), cellsInVolume, model, (char *)__FILE__, __LINE__ );

    /* stop inner timer */
    tend_inner = dtime() - tstart_inner;

    /* stop timer and compute statistics */
    safe_fclose ( modelname, model, (char *)__FILE__, __LINE__ );
    tend_outer = dtime() - tstart_outer;

    iospeed_inner = (bytesForVolume / (1000.f * 1000.f)) / tend_inner;
    iospeed_outer = (bytesForVolume / (1000.f * 1000.f)) / tend_outer;

    print_stats("Initial velocity model loaded (%lf GB)", TOGB(1.f * bytesForVolume));
    print_stats("\tInner time %lf seconds (%lf MiB/s)", tend_inner, iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MiB/s)", tend_outer, iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);

#endif /* end of DDO_NOT_PERFORM_IO clause */
};

/*
 * Saves the complete velocity field to disk.
 */
void write_snapshot(char *folder,
                    int suffix,
                    v_t *v,
                    const integer dimmz,
										const integer dimmx,
										const integer dimmy)
{
#if defined(DO_NOT_PERFORM_IO)
    print_debug("We are not writing the snapshot here cause IO is not enabled!");
#else
    /* local variables */
    double tstart_outer, tstart_inner;
    double iospeed_outer, iospeed_inner;
    double tend_outer, tend_inner;
		const  integer cellsInVolume = dimmz * dimmx * dimmy;
    char fname[300];
    int  rank = 0;

#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
#endif
    
		/* open snapshot file and write results */
    sprintf(fname,"%s/snapshot.%03d.%05d", folder, rank, suffix);
		
		print_debug("[Rank %d] is writting snapshot on %s", rank, fname);

    tstart_outer = dtime();
    FILE *snapshot = safe_fopen(fname,(char *)"wb", (char *)__FILE__, __LINE__ );


    tstart_inner = dtime();
    safe_fwrite( v->tr.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->tr.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->tr.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fwrite( v->tl.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->tl.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->tl.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fwrite( v->br.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->br.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->br.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fwrite( v->bl.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->bl.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fwrite( v->bl.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    /* stop inner timer */
    tend_inner = dtime();

    /* close file and stop outer timer */
    safe_fclose(fname, snapshot, (char *)__FILE__, __LINE__ );
    tend_outer = dtime();

    iospeed_inner = (( cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / (tend_inner - tstart_inner);
    iospeed_outer = (( cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / (tend_outer - tstart_outer);

    print_stats("Write snapshot (%lf GB)", TOGB(cellsInVolume * sizeof(real) * 12));
    print_stats("\tInner time %lf seconds (%lf MB/s)", (tend_inner - tstart_inner), iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MB/s)", (tend_outer - tstart_outer), iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);

#endif
};

/*
 * Reads the complete velocity field from disk.
 */
void read_snapshot(char *folder,
                   int suffix,
                   v_t *v,
                   const integer dimmz,
									 const integer dimmx,
									 const integer dimmy)
{
#if defined(DO_NOT_PERFORM_IO)
    print_debug("We are not reading the snapshot here cause IO is not enabled!");
#else
    /* local variables */
    double tstart_outer, tstart_inner;
    double iospeed_outer, iospeed_inner;
    double tend_outer, tend_inner;
		const integer cellsInVolume = dimmz * dimmx * dimmy;
    char fname[300];
		int  rank = 0;

#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
#endif
    
		/* open snapshot file and read results */
    sprintf(fname,"%s/snapshot.%03d.%05d", folder, rank, suffix);

		print_debug("[Rank %d] is freading snapshot from %s", rank, fname);
    
		tstart_outer = dtime();
    FILE *snapshot = safe_fopen(fname,(char *)"rb", (char *)__FILE__, __LINE__ );

    tstart_inner = dtime();
    safe_fread( v->tr.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->tr.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->tr.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fread( v->tl.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->tl.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->tl.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fread( v->br.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->br.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->br.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    safe_fread( v->bl.u, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->bl.v, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );
    safe_fread( v->bl.w, sizeof(real), cellsInVolume, snapshot, (char *)__FILE__, __LINE__ );

    /* stop inner timer */
    tend_inner = dtime() - tstart_inner;

    /* close file and stop outer timer */
    safe_fclose(fname, snapshot, (char *)__FILE__, __LINE__ );
    tend_outer = dtime() - tstart_outer;

    iospeed_inner = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / tend_inner;
    iospeed_outer = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / tend_outer;

    print_stats("Read snapshot (%lf GB)", TOGB(cellsInVolume * sizeof(real) * 12));
    print_stats("\tInner time %lf seconds (%lf MiB/s)", tend_inner, iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MiB/s)", tend_outer, iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);
#endif
};

void propagate_shot(time_d        direction,
                    v_t           v,
                    s_t           s,
                    coeff_t       coeffs,
                    real          *rho,
                    int           timesteps,
                    int           ntbwd,
                    real          dt,
                    real          dzi,
                    real          dxi,
                    real          dyi,
                    integer       nz0,
                    integer       nzf,
                    integer       nx0,
                    integer       nxf,
                    integer       ny0,
                    integer       nyf,
                    integer       stacki,
                    char          *folder,
                    real          *UNUSED(dataflush),
                    integer       dimmz,
                    integer       dimmx,
                    integer       dimmy)
{
    double tglobal_start, tglobal_total = 0.0;
    double tstress_start, tstress_total = 0.0;
    double tvel_start, tvel_total = 0.0;
    double megacells = 0.0;

		int rank=0, ranksize=1;

		#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    	MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    	MPI_Comm_size( MPI_COMM_WORLD, &ranksize);
		#endif


	for(int t=0; t < timesteps; t++)
    {
				/* print out some information */
        // print_info("[Rank %d]  Computing %d-th timestep", rank, t);

        /* perform IO */
        if ( t%stacki == 0 && direction == BACKWARD) read_snapshot(folder, ntbwd-t, &v, dimmz, dimmx, dimmy);

        tglobal_start = dtime();

        /* ------------------------------------------------------------------------------ */
        /*                      VELOCITY COMPUTATION                                      */
        /* ------------------------------------------------------------------------------ */
        /* Phase 1. Computation of the left-most planes of the domain */
        velocity_propagator(v, s, coeffs, rho, dt, dzi, dxi, dyi,
                            nz0 +   HALO,
                            nzf -   HALO,
                            nx0 +   HALO,
                            nxf -   HALO,
                            ny0 +   HALO,
                            ny0 + 2*HALO,
                            dimmz, dimmx );

        /* Phase 1. Computation of the right-most planes of the domain */
        velocity_propagator(v, s, coeffs, rho, dt, dzi, dxi, dyi,
                            nz0 +   HALO,
                            nzf -   HALO,
                            nx0 +   HALO,
                            nxf -   HALO,
                            nyf - 2*HALO,
                            nyf -   HALO,
                            dimmz, dimmx );
 
        /* Boundary exchange for velocity values */
        exchange_velocity_boundaries( v, dimmz * dimmx, rank, ranksize, nyf, ny0);
   
        /* Phase2. Computation of the central planes. */
        tvel_start = dtime();

        velocity_propagator(v, s, coeffs, rho, dt, dzi, dxi, dyi,
                            nz0 +   HALO,
                            nzf -   HALO,
                            nx0 +   HALO,
                            nxf -   HALO,
                            ny0 +   HALO,
                            nyf -   HALO,
                            dimmz, dimmx );
	
	tvel_total += (dtime() - tvel_start);

       
        /* ------------------------------------------------------------------------------ */
        /*                        STRESS COMPUTATION                                      */
        /* ------------------------------------------------------------------------------ */
        /* Phase 1. Computation of the left-most planes of the domain */
				stress_propagator(s, v, coeffs, rho, dt, dzi, dxi, dyi, 
                          nz0 +   HALO,
                          nzf -   HALO,
                          nx0 +   HALO,
                          nxf -   HALO,
                          ny0 +   HALO,
                          ny0 + 2*HALO,
                          dimmz, dimmx );

        /* Phase 1. Computation of the right-most planes of the domain */
				stress_propagator(s, v, coeffs, rho, dt, dzi, dxi, dyi, 
                          nz0 +   HALO,
                          nzf -   HALO,
                          nx0 +   HALO,
                          nxf -   HALO,
                          nyf - 2*HALO,
                          nyf -   HALO,
                          dimmz, dimmx );

        /* Boundary exchange for stress values */
        exchange_stress_boundaries( s, dimmz * dimmx, rank, ranksize, nyf, ny0);

        /* Phase 2 computation. Central planes of the domain (maingrid) */
        tstress_start = dtime();
        stress_propagator(s, v, coeffs, rho, dt, dzi, dxi, dyi, 
                          nz0 +   HALO,
                          nzf -   HALO,
                          nx0 +   HALO,
                          nxf -   HALO,
                          ny0 +   HALO,
                          nyf -   HALO,
                          dimmz, dimmx );
        
	tstress_total += (dtime() - tstress_start);
        tglobal_total += (dtime() - tglobal_start);

        /* perform IO */
        if ( t%stacki == 0 && direction == FORWARD) write_snapshot(folder, ntbwd-t, &v, dimmz, dimmx, dimmy);
    }
    
    /* compute some statistics */
    megacells = ((nzf - nz0) * (nxf - nx0) * (nyf - ny0)) / 1e6;
    tglobal_total /= (double) timesteps;
    tstress_total /= (double) timesteps;
    tvel_total    /= (double) timesteps;
    
    print_stats("Maingrid GLOBAL   computation took %lf seconds - %lf Mcells/s", tglobal_total, (2*megacells) / tglobal_total);
    print_stats("Maingrid STRESS   computation took %lf seconds - %lf Mcells/s", tstress_total,  megacells / tstress_total); 
    print_stats("Maingrid VELOCITY computation took %lf seconds - %lf Mcells/s", tvel_total, megacells / tvel_total); 
};

/* --------------- BOUNDARY EXCHANGES ---------------------------------------- */
void exchange_velocity_boundaries ( v_t v, 
                                    const integer plane_size, 
                                    const integer rank,
                                    const integer nranks,
                                    const integer nyf, 
                                    const integer ny0 )
{
    const integer num_planes = HALO;
    const integer nelems     = num_planes * plane_size;

    const integer left_recv  = ny0;
    const integer left_send  = ny0+HALO;

    const integer right_recv = nyf-HALO;
    const integer right_send = nyf-2*HALO;
    
    if ( rank != 0 )
    {
        // [RANK-1] <---> [RANK] communication
        EXCHANGE( &v.tl.u[left_send], &v.tl.u[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tl.v[left_send], &v.tl.v[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tl.w[left_send], &v.tl.w[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.tr.u[left_send], &v.tr.u[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tr.v[left_send], &v.tr.v[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tr.w[left_send], &v.tr.w[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.bl.u[left_send], &v.bl.u[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.bl.v[left_send], &v.bl.v[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.bl.w[left_send], &v.bl.w[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.br.u[left_send], &v.br.u[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.br.v[left_send], &v.br.v[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.br.w[left_send], &v.br.w[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
    }

    if ( rank != nranks -1 )  //task to exchange stress boundaries
    {
        //                [RANK] <---> [RANK+1] communication
        EXCHANGE( &v.tl.u[right_send], &v.tl.u[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tl.v[right_send], &v.tl.v[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tl.w[right_send], &v.tl.w[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.tr.u[right_send], &v.tr.u[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tr.v[right_send], &v.tr.v[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.tr.w[right_send], &v.tr.w[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.bl.u[right_send], &v.bl.u[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.bl.v[right_send], &v.bl.v[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.bl.w[right_send], &v.bl.w[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &v.br.u[right_send], &v.br.u[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.br.v[right_send], &v.br.v[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &v.br.w[right_send], &v.br.w[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
    }

		print_debug("Velocity boundaries exchanged correctly");
};

void exchange_stress_boundaries ( s_t s, 
                                  const integer plane_size, 
                                  const integer rank,
                                  const integer nranks,
                                  const integer nyf, 
                                  const integer ny0 )
{
    const integer num_planes = HALO;
    const integer nelems     = num_planes * plane_size;

    const integer left_recv  = ny0;
    const integer left_send  = ny0+HALO;

    const integer right_recv = nyf-HALO;
    const integer right_send = nyf-2*HALO;

    if ( rank != 0 )
    {
        // [RANK-1] <---> [RANK] communication
        EXCHANGE( &s.tl.zz[left_send], &s.tl.zz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xz[left_send], &s.tl.xz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.yz[left_send], &s.tl.yz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xx[left_send], &s.tl.xx[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xy[left_send], &s.tl.xy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.yy[left_send], &s.tl.yy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.tr.zz[left_send], &s.tr.zz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xz[left_send], &s.tr.xz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.yz[left_send], &s.tr.yz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xx[left_send], &s.tr.xx[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xy[left_send], &s.tr.xy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.yy[left_send], &s.tr.yy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.bl.zz[left_send], &s.bl.zz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xz[left_send], &s.bl.xz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.yz[left_send], &s.bl.yz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xx[left_send], &s.bl.xx[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xy[left_send], &s.bl.xy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.yy[left_send], &s.bl.yy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.br.zz[left_send], &s.br.zz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xz[left_send], &s.br.xz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.yz[left_send], &s.br.yz[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xx[left_send], &s.br.xx[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xy[left_send], &s.br.xy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.yy[left_send], &s.br.yy[left_recv], rank-1, rank, nelems, __FILE__, __LINE__ );
    }
    
    if ( rank != nranks-1 )
    {
        //                [RANK] <---> [RANK+1] communication
        EXCHANGE( &s.tl.zz[right_send], &s.tl.zz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xz[right_send], &s.tl.xz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.yz[right_send], &s.tl.yz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xx[right_send], &s.tl.xx[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.xy[right_send], &s.tl.xy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tl.yy[right_send], &s.tl.yy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.tr.zz[right_send], &s.tr.zz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xz[right_send], &s.tr.xz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.yz[right_send], &s.tr.yz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xx[right_send], &s.tr.xx[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.xy[right_send], &s.tr.xy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.tr.yy[right_send], &s.tr.yy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.bl.zz[right_send], &s.bl.zz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xz[right_send], &s.bl.xz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.yz[right_send], &s.bl.yz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xx[right_send], &s.bl.xx[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.xy[right_send], &s.bl.xy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.bl.yy[right_send], &s.bl.yy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );

        EXCHANGE( &s.br.zz[right_send], &s.br.zz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xz[right_send], &s.br.xz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.yz[right_send], &s.br.yz[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xx[right_send], &s.br.xx[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.xy[right_send], &s.br.xy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
        EXCHANGE( &s.br.yy[right_send], &s.br.yy[right_recv], rank+1, rank, nelems, __FILE__, __LINE__ );
    }

		print_debug("Stress boundaries exchanged correctly");
};

void EXCHANGE (const real*   sendbuf, 
                          real*   recvbuf, 
                    const integer dst, 
                    const integer src, 
                    const integer message_size,
                    const char*   file,
                    const integer line)
{	
#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
 	 int err;
   int tag = 100;
   
   print_debug( "         [BEFORE]MPI sendrecv [count:%d][dst:%d][src:%d] %s : %d", 
 			message_size,  dst, src, file, line);

   MPI_Status  statuses[2];
   MPI_Request requests[2];
   
   MPI_Irecv( recvbuf, message_size, MPI_FLOAT, dst, tag, MPI_COMM_WORLD, &requests[0] );
   MPI_Isend( sendbuf, message_size, MPI_FLOAT, dst, tag, MPI_COMM_WORLD, &requests[1] );
   err = MPI_Waitall(2, requests, statuses);

   print_debug( "         [AFTER ]MPI sendrecv                          %s : %d", 
 			file, line);    

   if ( err != MPI_SUCCESS )
 	{
 		print_error("MPI error %d!", err);
 		abort();
 	}
	
#endif
};

