#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  fwi_kernel.c
 *
 *    Description:  kernel propagator implementation
 *
 *        Version:  1.0
 *        Created:  14/12/15 12:10:05
 *       Revision:  none
 *       Compiler:  icc
 *
 *         Author:  YOUR NAME (),
 *   Organization:
 *
 * =====================================================================================
 */

#include "fwi_kernel.cuh"


/*
 * Initializes an array of length "length" to a random number.
 */

void write_velocity_datafile(v_t     *v,
                             s_t     *s,
                             coeff_t *c,
                             real    *rho,
                             const integer dimmx,
                             const integer dimmy,
                             const integer dimmz){
    char debug[300];
    const integer cellsInVolume = dimmz * dimmx * dimmy;

    sprintf( debug, "./velocityData.txt");

    FILE *debugFile=fopen(debug,"wb");

    fwrite(v->tl.w, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->tr.w, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->bl.w, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->br.w, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->tl.u, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->tr.u, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->bl.u, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->br.u, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->tl.v, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->tr.v, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->bl.v, sizeof(real), cellsInVolume,debugFile );
    fwrite(v->br.v, sizeof(real), cellsInVolume,debugFile );

    fwrite(s->bl.zz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.zz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.zz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tr.zz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->bl.xz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.xz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.xz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tr.xz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->bl.yz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.yz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.yz, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tr.yz, sizeof(real), cellsInVolume,debugFile );

    fwrite(s->tr.xx, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.xx, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.xx, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->bl.xx, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tr.xy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.xy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.xy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->bl.xy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tl.yy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->tr.yy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->bl.yy, sizeof(real), cellsInVolume,debugFile );
    fwrite(s->br.yy, sizeof(real), cellsInVolume,debugFile );

    fclose(debugFile);
}



//__global__
void
set_array_to_random_real(real *__restrict__ array, const integer length) {
    const real randvalue = rand() / (1.0 * RAND_MAX);

//
//    int i = blockDim.x *
//            blockIdx.x +
//            threadIdx.x;
//
//    if (i < length)
//    {
//        array[i]= randvalue;
//    }
    for (integer i = 0; i < length; i++)
        array[i] = randvalue;

}

/*
 * Initializes an array of length "length" to a constant floating point value.
 */
//__global__
void set_array_to_constant(real *__restrict__ array, const real value, const integer length) {
//    int i = blockDim.x *
//            blockIdx.x +
//            threadIdx.x;
//
//    if (i < length)
//    {
//        array[i]= value;
//    }

    for (integer i = 0; i < length; i++)
        array[i] = value;
}

void check_memory_shot(const integer dimmz,
                       const integer dimmx,
                       const integer dimmy,
                       coeff_t *c,
                       s_t *s,
                       v_t *v,
                       real *rho) {
#if defined(DEBUG)
    print_debug("Checking memory shot values");

    real UNUSED(value);
    for( int i=0; i < (dimmz * dimmx * dimmy); i++)
    {
        value = c->c11[i];
        value = c->c12[i];
        value = c->c13[i];
        value = c->c14[i];
        value = c->c15[i];
        value = c->c16[i];

        value = c->c22[i];
        value = c->c23[i];
        value = c->c24[i];
        value = c->c25[i];
        value = c->c26[i];

        value = c->c33[i];
        value = c->c34[i];
        value = c->c35[i];
        value = c->c36[i];

        value = c->c44[i];
        value = c->c45[i];
        value = c->c46[i];
        
        value = c->c55[i];
        value = c->c56[i];
        value = c->c66[i];

        value = v->tl.u[i];
        value = v->tl.v[i];
        value = v->tl.w[i];

        value = v->tr.u[i];
        value = v->tr.v[i];
        value = v->tr.w[i];

        value = v->bl.u[i];
        value = v->bl.v[i];
        value = v->bl.w[i];
        
        value = v->br.u[i];
        value = v->br.v[i];
        value = v->br.w[i];

        value = rho[i];
    }
        print_debug("Shot memory is well allocated");
#endif
};


void alloc_memory_shot_gpu(const integer dimmz,
                           const integer dimmx,
                           const integer dimmy,
                           coeff_t *gpu_c,
                           s_t *gpu_s,
                           v_t *gpu_v,
                           real **gpu_rho) {
    const size_t size = dimmz * dimmx * dimmy * sizeof(real);

    hipMalloc(&(gpu_c->c11), size);
    hipMalloc(&(gpu_c->c12), size);
    hipMalloc(&(gpu_c->c13), size);
    hipMalloc(&(gpu_c->c14), size);
    hipMalloc(&(gpu_c->c15), size);
    hipMalloc(&(gpu_c->c16), size);

    hipMalloc(&(gpu_c->c22), size);
    hipMalloc(&(gpu_c->c23), size);
    hipMalloc(&(gpu_c->c24), size);
    hipMalloc(&(gpu_c->c25), size);
    hipMalloc(&(gpu_c->c26), size);

    hipMalloc(&(gpu_c->c33), size);
    hipMalloc(&(gpu_c->c34), size);
    hipMalloc(&(gpu_c->c35), size);
    hipMalloc(&(gpu_c->c36), size);

    hipMalloc(&(gpu_c->c44), size);
    hipMalloc(&(gpu_c->c45), size);
    hipMalloc(&(gpu_c->c46), size);

    hipMalloc(&(gpu_c->c55), size);
    hipMalloc(&(gpu_c->c56), size);
    hipMalloc(&(gpu_c->c66), size);

    hipMalloc(&(gpu_v->tl.u), size);
    hipMalloc(&(gpu_v->tl.v), size);
    hipMalloc(&(gpu_v->tl.w), size);

    hipMalloc(&(gpu_v->tr.u), size);
    hipMalloc(&(gpu_v->tr.v), size);
    hipMalloc(&(gpu_v->tr.w), size);

    hipMalloc(&(gpu_v->bl.u), size);
    hipMalloc(&(gpu_v->bl.v), size);
    hipMalloc(&(gpu_v->bl.w), size);

    hipMalloc(&(gpu_v->br.u), size);
    hipMalloc(&(gpu_v->br.v), size);
    hipMalloc(&(gpu_v->br.w), size);

    hipMalloc(&(gpu_s->tl.zz), size);
    hipMalloc(&(gpu_s->tl.xz), size);
    hipMalloc(&(gpu_s->tl.yz), size);
    hipMalloc(&(gpu_s->tl.xx), size);
    hipMalloc(&(gpu_s->tl.xy), size);
    hipMalloc(&(gpu_s->tl.yy), size);

    hipMalloc(&(gpu_s->tr.zz), size);
    hipMalloc(&(gpu_s->tr.xz), size);
    hipMalloc(&(gpu_s->tr.yz), size);
    hipMalloc(&(gpu_s->tr.xx), size);
    hipMalloc(&(gpu_s->tr.xy), size);
    hipMalloc(&(gpu_s->tr.yy), size);

    hipMalloc(&(gpu_s->bl.zz), size);
    hipMalloc(&(gpu_s->bl.xz), size);
    hipMalloc(&(gpu_s->bl.yz), size);
    hipMalloc(&(gpu_s->bl.xx), size);
    hipMalloc(&(gpu_s->bl.xy), size);
    hipMalloc(&(gpu_s->bl.yy), size);

    hipMalloc(&(gpu_s->br.zz), size);
    hipMalloc(&(gpu_s->br.xz), size);
    hipMalloc(&(gpu_s->br.yz), size);
    hipMalloc(&(gpu_s->br.xx), size);
    hipMalloc(&(gpu_s->br.xy), size);
    hipMalloc(&(gpu_s->br.yy), size);

    hipMalloc(&(*gpu_rho), size);

}

void alloc_memory_shot(const integer dimmz,
                       const integer dimmx,
                       const integer dimmy,
                       coeff_t *c,
                       s_t *s,
                       v_t *v,
                       real **rho
) {
    const size_t size = dimmz * dimmx * dimmy * sizeof(real);

    print_debug("ptr size = %lu bytes (%lu elements)",
                size,
                (size_t) dimmz * dimmx * dimmy);

    /* allocate coefficients */
    c->c11 = (real *) __malloc(ALIGN_REAL, size);
    c->c12 = (real *) __malloc(ALIGN_REAL, size);
    c->c13 = (real *) __malloc(ALIGN_REAL, size);
    c->c14 = (real *) __malloc(ALIGN_REAL, size);
    c->c15 = (real *) __malloc(ALIGN_REAL, size);
    c->c16 = (real *) __malloc(ALIGN_REAL, size);

    c->c22 = (real *) __malloc(ALIGN_REAL, size);
    c->c23 = (real *) __malloc(ALIGN_REAL, size);
    c->c24 = (real *) __malloc(ALIGN_REAL, size);
    c->c25 = (real *) __malloc(ALIGN_REAL, size);
    c->c26 = (real *) __malloc(ALIGN_REAL, size);

    c->c33 = (real *) __malloc(ALIGN_REAL, size);
    c->c34 = (real *) __malloc(ALIGN_REAL, size);
    c->c35 = (real *) __malloc(ALIGN_REAL, size);
    c->c36 = (real *) __malloc(ALIGN_REAL, size);

    c->c44 = (real *) __malloc(ALIGN_REAL, size);
    c->c45 = (real *) __malloc(ALIGN_REAL, size);
    c->c46 = (real *) __malloc(ALIGN_REAL, size);

    c->c55 = (real *) __malloc(ALIGN_REAL, size);
    c->c56 = (real *) __malloc(ALIGN_REAL, size);
    c->c66 = (real *) __malloc(ALIGN_REAL, size);

    /* allocate velocity components */
    v->tl.u = (real *) __malloc(ALIGN_REAL, size);
    v->tl.v = (real *) __malloc(ALIGN_REAL, size);
    v->tl.w = (real *) __malloc(ALIGN_REAL, size);

    v->tr.u = (real *) __malloc(ALIGN_REAL, size);
    v->tr.v = (real *) __malloc(ALIGN_REAL, size);
    v->tr.w = (real *) __malloc(ALIGN_REAL, size);

    v->bl.u = (real *) __malloc(ALIGN_REAL, size);
    v->bl.v = (real *) __malloc(ALIGN_REAL, size);
    v->bl.w = (real *) __malloc(ALIGN_REAL, size);

    v->br.u = (real *) __malloc(ALIGN_REAL, size);
    v->br.v = (real *) __malloc(ALIGN_REAL, size);
    v->br.w = (real *) __malloc(ALIGN_REAL, size);

    /* allocate stress components   */
    s->tl.zz = (real *) __malloc(ALIGN_REAL, size);
    s->tl.xz = (real *) __malloc(ALIGN_REAL, size);
    s->tl.yz = (real *) __malloc(ALIGN_REAL, size);
    s->tl.xx = (real *) __malloc(ALIGN_REAL, size);
    s->tl.xy = (real *) __malloc(ALIGN_REAL, size);
    s->tl.yy = (real *) __malloc(ALIGN_REAL, size);

    s->tr.zz = (real *) __malloc(ALIGN_REAL, size);
    s->tr.xz = (real *) __malloc(ALIGN_REAL, size);
    s->tr.yz = (real *) __malloc(ALIGN_REAL, size);
    s->tr.xx = (real *) __malloc(ALIGN_REAL, size);
    s->tr.xy = (real *) __malloc(ALIGN_REAL, size);
    s->tr.yy = (real *) __malloc(ALIGN_REAL, size);

    s->bl.zz = (real *) __malloc(ALIGN_REAL, size);
    s->bl.xz = (real *) __malloc(ALIGN_REAL, size);
    s->bl.yz = (real *) __malloc(ALIGN_REAL, size);
    s->bl.xx = (real *) __malloc(ALIGN_REAL, size);
    s->bl.xy = (real *) __malloc(ALIGN_REAL, size);
    s->bl.yy = (real *) __malloc(ALIGN_REAL, size);

    s->br.zz = (real *) __malloc(ALIGN_REAL, size);
    s->br.xz = (real *) __malloc(ALIGN_REAL, size);
    s->br.yz = (real *) __malloc(ALIGN_REAL, size);
    s->br.xx = (real *) __malloc(ALIGN_REAL, size);
    s->br.xy = (real *) __malloc(ALIGN_REAL, size);
    s->br.yy = (real *) __malloc(ALIGN_REAL, size);

    /* allocate density array       */
    *rho = (real *) __malloc(ALIGN_REAL, size);
};

void free_memory_shot_gpu(coeff_t *gpu_c,
                          s_t *gpu_s,
                          v_t *gpu_v,
                          real **gpu_rho) {
    hipFree((void *) gpu_c->c11);
    hipFree((void *) gpu_c->c12);
    hipFree((void *) gpu_c->c13);
    hipFree((void *) gpu_c->c14);
    hipFree((void *) gpu_c->c15);
    hipFree((void *) gpu_c->c16);

    hipFree((void *) gpu_c->c22);
    hipFree((void *) gpu_c->c23);
    hipFree((void *) gpu_c->c24);
    hipFree((void *) gpu_c->c25);
    hipFree((void *) gpu_c->c26);
    hipFree((void *) gpu_c->c33);
    hipFree((void *) gpu_c->c34);
    hipFree((void *) gpu_c->c35);
    hipFree((void *) gpu_c->c36);

    hipFree((void *) gpu_c->c44);
    hipFree((void *) gpu_c->c45);
    hipFree((void *) gpu_c->c46);

    hipFree((void *) gpu_c->c55);
    hipFree((void *) gpu_c->c56);

    hipFree((void *) gpu_c->c66);

    hipFree((void *) gpu_v->tl.u);
    hipFree((void *) gpu_v->tl.v);
    hipFree((void *) gpu_v->tl.w);

    hipFree((void *) gpu_v->tr.u);
    hipFree((void *) gpu_v->tr.v);
    hipFree((void *) gpu_v->tr.w);

    hipFree((void *) gpu_v->bl.u);
    hipFree((void *) gpu_v->bl.v);
    hipFree((void *) gpu_v->bl.w);

    hipFree((void *) gpu_v->br.u);
    hipFree((void *) gpu_v->br.v);
    hipFree((void *) gpu_v->br.w);

    hipFree((void *) gpu_s->tl.zz);
    hipFree((void *) gpu_s->tl.xz);
    hipFree((void *) gpu_s->tl.yz);
    hipFree((void *) gpu_s->tl.xx);
    hipFree((void *) gpu_s->tl.xy);
    hipFree((void *) gpu_s->tl.yy);

    hipFree((void *) gpu_s->tr.zz);
    hipFree((void *) gpu_s->tr.xz);
    hipFree((void *) gpu_s->tr.yz);
    hipFree((void *) gpu_s->tr.xx);
    hipFree((void *) gpu_s->tr.xy);
    hipFree((void *) gpu_s->tr.yy);

    hipFree((void *) gpu_s->bl.zz);
    hipFree((void *) gpu_s->bl.xz);
    hipFree((void *) gpu_s->bl.yz);
    hipFree((void *) gpu_s->bl.xx);
    hipFree((void *) gpu_s->bl.xy);
    hipFree((void *) gpu_s->bl.yy);

    hipFree((void *) gpu_s->br.zz);
    hipFree((void *) gpu_s->br.xz);
    hipFree((void *) gpu_s->br.yz);
    hipFree((void *) gpu_s->br.xx);
    hipFree((void *) gpu_s->br.xy);
    hipFree((void *) gpu_s->br.yy);

    hipFree((void *) *gpu_rho);

}

void free_memory_shot(coeff_t *c,
                      s_t *s,
                      v_t *v,
                      real **rho) {
    /* deallocate coefficients */
    __free((void *) c->c11);
    __free((void *) c->c12);
    __free((void *) c->c13);
    __free((void *) c->c14);
    __free((void *) c->c15);
    __free((void *) c->c16);

    __free((void *) c->c22);
    __free((void *) c->c23);
    __free((void *) c->c24);
    __free((void *) c->c25);
    __free((void *) c->c26);
    __free((void *) c->c33);

    __free((void *) c->c34);
    __free((void *) c->c35);
    __free((void *) c->c36);

    __free((void *) c->c44);
    __free((void *) c->c45);
    __free((void *) c->c46);

    __free((void *) c->c55);
    __free((void *) c->c56);

    __free((void *) c->c66);

    __free((void *) v->tl.u);
    __free((void *) v->tl.v);
    __free((void *) v->tl.w);

    __free((void *) v->tr.u);
    __free((void *) v->tr.v);
    __free((void *) v->tr.w);

    __free((void *) v->bl.u);
    __free((void *) v->bl.v);
    __free((void *) v->bl.w);

    __free((void *) v->br.u);
    __free((void *) v->br.v);
    __free((void *) v->br.w);

    __free((void *) s->tl.zz);
    __free((void *) s->tl.xz);
    __free((void *) s->tl.yz);
    __free((void *) s->tl.xx);
    __free((void *) s->tl.xy);
    __free((void *) s->tl.yy);

    __free((void *) s->tr.zz);
    __free((void *) s->tr.xz);
    __free((void *) s->tr.yz);
    __free((void *) s->tr.xx);
    __free((void *) s->tr.xy);
    __free((void *) s->tr.yy);

    __free((void *) s->bl.zz);
    __free((void *) s->bl.xz);
    __free((void *) s->bl.yz);
    __free((void *) s->bl.xx);
    __free((void *) s->bl.xy);
    __free((void *) s->bl.yy);

    __free((void *) s->br.zz);
    __free((void *) s->br.xz);
    __free((void *) s->br.yz);
    __free((void *) s->br.xx);
    __free((void *) s->br.xy);
    __free((void *) s->br.yy);


    /* deallocate density array       */
    __free((void *) *rho);
};

/*
 * Loads initial values from coeffs, stress and velocity.
 *
 * dimmz: number of z planes.
 * dimmx: number of x planes
 * FirstYPlane: first Y plane of my local domain (includes HALO)
 * LastYPlane: last Y plane of my local domain (includes HALO)
 */
void load_local_velocity_model(const real waveletFreq,
                               const integer dimmz,
                               const integer dimmx,
                               const integer FirstYPlane,
                               const integer LastYPlane,
                               coeff_t *c,
                               s_t *s,
                               v_t *v,
                               real *rho) {
    /* Local variables */
    double tstart_outer, tstart_inner, tend_outer, tend_inner;
    double iospeed_inner, iospeed_outer;
    char modelname[300];

    const integer cellsInVolume = dimmz * dimmx * (LastYPlane - FirstYPlane);
    const integer bytesForVolume = WRITTEN_FIELDS * cellsInVolume * sizeof(real);

    /*
     * Material, velocities and stresses are initizalized
     * accorting to the compilation flags, either randomly
     * or by reading an input velocity model.
     */

    /* initialize stress arrays */
    set_array_to_constant(s->tl.zz, 0, cellsInVolume);
    set_array_to_constant(s->tl.xz, 0, cellsInVolume);
    set_array_to_constant(s->tl.yz, 0, cellsInVolume);
    set_array_to_constant(s->tl.xx, 0, cellsInVolume);
    set_array_to_constant(s->tl.xy, 0, cellsInVolume);
    set_array_to_constant(s->tl.yy, 0, cellsInVolume);
    set_array_to_constant(s->tr.zz, 0, cellsInVolume);
    set_array_to_constant(s->tr.xz, 0, cellsInVolume);
    set_array_to_constant(s->tr.yz, 0, cellsInVolume);
    set_array_to_constant(s->tr.xx, 0, cellsInVolume);
    set_array_to_constant(s->tr.xy, 0, cellsInVolume);
    set_array_to_constant(s->tr.yy, 0, cellsInVolume);
    set_array_to_constant(s->bl.zz, 0, cellsInVolume);
    set_array_to_constant(s->bl.xz, 0, cellsInVolume);
    set_array_to_constant(s->bl.yz, 0, cellsInVolume);
    set_array_to_constant(s->bl.xx, 0, cellsInVolume);
    set_array_to_constant(s->bl.xy, 0, cellsInVolume);
    set_array_to_constant(s->bl.yy, 0, cellsInVolume);
    set_array_to_constant(s->br.zz, 0, cellsInVolume);
    set_array_to_constant(s->br.xz, 0, cellsInVolume);
    set_array_to_constant(s->br.yz, 0, cellsInVolume);
    set_array_to_constant(s->br.xx, 0, cellsInVolume);
    set_array_to_constant(s->br.xy, 0, cellsInVolume);
    set_array_to_constant(s->br.yy, 0, cellsInVolume);

#if defined(DO_NOT_PERFORM_IO)

    /* initialize material coefficients */
    set_array_to_random_real( c->c11, cellsInVolume);
    set_array_to_random_real( c->c12, cellsInVolume);
    set_array_to_random_real( c->c13, cellsInVolume);
    set_array_to_random_real( c->c14, cellsInVolume);
    set_array_to_random_real( c->c15, cellsInVolume);
    set_array_to_random_real( c->c16, cellsInVolume);
    set_array_to_random_real( c->c22, cellsInVolume);
    set_array_to_random_real( c->c23, cellsInVolume);
    set_array_to_random_real( c->c24, cellsInVolume);
    set_array_to_random_real( c->c25, cellsInVolume);
    set_array_to_random_real( c->c26, cellsInVolume);
    set_array_to_random_real( c->c33, cellsInVolume);
    set_array_to_random_real( c->c34, cellsInVolume);
    set_array_to_random_real( c->c35, cellsInVolume);
    set_array_to_random_real( c->c36, cellsInVolume);
    set_array_to_random_real( c->c44, cellsInVolume);
    set_array_to_random_real( c->c45, cellsInVolume);
    set_array_to_random_real( c->c46, cellsInVolume);
    set_array_to_random_real( c->c55, cellsInVolume);
    set_array_to_random_real( c->c56, cellsInVolume);
    set_array_to_random_real( c->c66, cellsInVolume);
    
    /* initalize velocity components */
    set_array_to_random_real( v->tl.u, cellsInVolume );
    set_array_to_random_real( v->tl.v, cellsInVolume );
    set_array_to_random_real( v->tl.w, cellsInVolume );
    set_array_to_random_real( v->tr.u, cellsInVolume );
    set_array_to_random_real( v->tr.v, cellsInVolume );
    set_array_to_random_real( v->tr.w, cellsInVolume );
    set_array_to_random_real( v->bl.u, cellsInVolume );
    set_array_to_random_real( v->bl.v, cellsInVolume );
    set_array_to_random_real( v->bl.w, cellsInVolume );
    set_array_to_random_real( v->br.u, cellsInVolume );
    set_array_to_random_real( v->br.v, cellsInVolume );
    set_array_to_random_real( v->br.w, cellsInVolume );

    /* initialize density (rho) */
    set_array_to_random_real( rho, cellsInVolume );

#else /* load velocity model from external file */

    /* initialize material coefficients */
    set_array_to_constant(c->c11, 1.0, cellsInVolume);
    set_array_to_constant(c->c12, 1.0, cellsInVolume);
    set_array_to_constant(c->c13, 1.0, cellsInVolume);
    set_array_to_constant(c->c14, 1.0, cellsInVolume);
    set_array_to_constant(c->c15, 1.0, cellsInVolume);
    set_array_to_constant(c->c16, 1.0, cellsInVolume);
    set_array_to_constant(c->c22, 1.0, cellsInVolume);
    set_array_to_constant(c->c23, 1.0, cellsInVolume);
    set_array_to_constant(c->c24, 1.0, cellsInVolume);
    set_array_to_constant(c->c25, 1.0, cellsInVolume);
    set_array_to_constant(c->c26, 1.0, cellsInVolume);
    set_array_to_constant(c->c33, 1.0, cellsInVolume);
    set_array_to_constant(c->c34, 1.0, cellsInVolume);
    set_array_to_constant(c->c35, 1.0, cellsInVolume);
    set_array_to_constant(c->c36, 1.0, cellsInVolume);
    set_array_to_constant(c->c44, 1.0, cellsInVolume);
    set_array_to_constant(c->c45, 1.0, cellsInVolume);
    set_array_to_constant(c->c46, 1.0, cellsInVolume);
    set_array_to_constant(c->c55, 1.0, cellsInVolume);
    set_array_to_constant(c->c56, 1.0, cellsInVolume);
    set_array_to_constant(c->c66, 1.0, cellsInVolume);

    /* initialize density (rho) */
    set_array_to_constant(rho, 1.0, cellsInVolume);

    char *fwipath = read_env_variable("FWIDIR");
    /* open initial model, binary file */
    sprintf(modelname, "%s/InputModels/velocitymodel_(%.2f).bin", fwipath, waveletFreq);
    print_info("Loading input model %s from disk (this could take a while)", modelname);

    /* start clock, take into account file opening */
    tstart_outer = dtime();
    FILE *model = safe_fopen(modelname, (char *) "rb", (char *) __FILE__, __LINE__);

    /* seek to the correct position corresponding to mpi_rank */
    fseek(model, sizeof(real) * WRITTEN_FIELDS * dimmz * dimmx * FirstYPlane, SEEK_SET);

    /* start clock, do not take into account file opening */
    tstart_inner = dtime();

    /* initalize velocity components */
    safe_fread(v->tl.u, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->tl.v, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->tl.w, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->tr.u, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->tr.v, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->tr.w, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->bl.u, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->bl.v, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->bl.w, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->br.u, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->br.v, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);
    safe_fread(v->br.w, sizeof(real), cellsInVolume, model, (char *) __FILE__, __LINE__);

    /* stop inner timer */
    tend_inner = dtime() - tstart_inner;

    /* stop timer and compute statistics */
    safe_fclose(modelname, model, (char *) __FILE__, __LINE__);
    tend_outer = dtime() - tstart_outer;

    iospeed_inner = (bytesForVolume / (1000.f * 1000.f)) / tend_inner;
    iospeed_outer = (bytesForVolume / (1000.f * 1000.f)) / tend_outer;

    print_stats("Initial velocity model loaded (%lf GB)", TOGB(1.f * bytesForVolume));
    print_stats("\tInner time %lf seconds (%lf MiB/s)", tend_inner, iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MiB/s)", tend_outer, iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);

#endif /* end of DDO_NOT_PERFORM_IO clause */
};

void copy_velocity_model_ToGpu(const integer dimmz,
                               const integer dimmx,
                               const integer FirstYPlane,
                               const integer LastYPlane,
                               coeff_t *c,
                               s_t *s,
                               v_t *v,
                               real *rho,
                               coeff_t *gpu_c,
                               s_t *gpu_s,
                               v_t *gpu_v,
                               real *gpu_rho) {

    const integer cellsInVolume = dimmz * dimmx * (LastYPlane - FirstYPlane);

    hipMemcpy(gpu_s->tl.zz, s->tl.zz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tl.xz, s->tl.xz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tl.yz, s->tl.yz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tl.xx, s->tl.xx, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tl.xy, s->tl.xy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tl.yy, s->tl.yy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.zz, s->tr.zz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.xz, s->tr.xz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.yz, s->tr.yz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.xx, s->tr.xx, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.xy, s->tr.xy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->tr.yy, s->tr.yy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.zz, s->bl.zz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.xz, s->bl.xz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.yz, s->bl.yz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.xx, s->bl.xx, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.xy, s->bl.xy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->bl.yy, s->bl.yy, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->br.zz, s->br.zz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->br.xz, s->br.xz, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_s->br.yz, s->br.yz, cellsInVolume, hipMemcpyHostToDevice);

    hipMemcpy(gpu_c->c11, c->c11, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c12, c->c12, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c13, c->c13, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c14, c->c14, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c15, c->c15, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c16, c->c16, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c22, c->c22, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c23, c->c23, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c24, c->c24, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c25, c->c25, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c26, c->c26, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c33, c->c33, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c34, c->c34, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c35, c->c35, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c36, c->c36, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c44, c->c44, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c45, c->c45, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c46, c->c46, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c55, c->c55, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c56, c->c56, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c->c66, c->c66, cellsInVolume, hipMemcpyHostToDevice);

    hipMemcpy(gpu_v->tl.u, v->tl.u, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->tl.v, v->tl.v, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->tl.w, v->tl.w, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->tr.u, v->tr.u, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->tr.v, v->tr.v, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->tr.w, v->tr.w, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->bl.u, v->bl.u, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->bl.v, v->bl.v, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->bl.w, v->bl.w, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->br.u, v->br.u, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->br.v, v->br.v, cellsInVolume, hipMemcpyHostToDevice);
    hipMemcpy(gpu_v->br.w, v->br.w, cellsInVolume, hipMemcpyHostToDevice);

    hipMemcpy(gpu_rho, rho, cellsInVolume, hipMemcpyHostToDevice);
}


void copy_velocity_data_ToCPU ( v_t     *v,
                                s_t     *s,
                                coeff_t *c,
                                real    *rho,
                                v_t     *gpu_v,
                                s_t     *gpu_s,
                                coeff_t *gpu_c,
                                real    *gpu_rho,
                                const integer dimmx,
                                const integer dimmy,
                                const integer dimmz
){
    const integer cellsInVolume = dimmz * dimmx * dimmy;

    hipMemcpy(v->tl.w,gpu_v->tl.w,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->tr.w,gpu_v->tr.w,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->bl.w,gpu_v->bl.w,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->br.w,gpu_v->br.w,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->tl.u,gpu_v->tl.u,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->tr.u,gpu_v->tr.u,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->bl.u,gpu_v->bl.u,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->br.u,gpu_v->br.u,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->tl.v,gpu_v->tl.v,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->tr.v,gpu_v->tr.v,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->bl.v,gpu_v->bl.v,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(v->br.v,gpu_v->br.v,  cellsInVolume, hipMemcpyDeviceToHost);

    hipMemcpy(s->bl.zz,gpu_s->bl.zz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.zz,gpu_s->br.zz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.zz,gpu_s->tl.zz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tr.zz,gpu_s->tr.zz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->bl.xz,gpu_s->bl.xz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.xz,gpu_s->br.xz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.xz,gpu_s->tl.xz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tr.xz,gpu_s->tr.xz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->bl.yz,gpu_s->bl.yz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.yz,gpu_s->br.yz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.yz,gpu_s->tl.yz,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tr.yz,gpu_s->tr.yz,  cellsInVolume, hipMemcpyDeviceToHost);


    hipMemcpy(s->tr.xx,gpu_s->tr.xx,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.xx,gpu_s->tl.xx,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.xx,gpu_s->br.xx,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->bl.xx,gpu_s->bl.xx,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tr.xy,gpu_s->tr.xy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.xy,gpu_s->tl.xy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.xy,gpu_s->br.xy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->bl.xy,gpu_s->bl.xy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tl.yy,gpu_s->tl.yy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->tr.yy,gpu_s->tr.yy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->bl.yy,gpu_s->bl.yy,  cellsInVolume, hipMemcpyDeviceToHost);
    hipMemcpy(s->br.yy,gpu_s->br.yy,  cellsInVolume, hipMemcpyDeviceToHost);
};

/*
 * Saves the complete velocity field to disk.
 */
void write_snapshot(char *folder,
                    int suffix,
                    v_t *v,
                    const integer dimmz,
                    const integer dimmx,
                    const integer dimmy) {
#if defined(DO_NOT_PERFORM_IO)
    print_debug("We are not writing the snapshot here cause IO is not enabled!");
#else
    /* local variables */
    double tstart_outer, tstart_inner;
    double iospeed_outer, iospeed_inner;
    double tend_outer, tend_inner;
    const integer cellsInVolume = dimmz * dimmx * dimmy;
    char fname[300];
    int rank = 0;

#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
#endif

    /* open snapshot file and write results */
    sprintf(fname, "%s/snapshot.%03d.%05d", folder, rank, suffix);

    print_debug("[Rank %d] is writting snapshot on %s", rank, fname);

    tstart_outer = dtime();
    FILE *snapshot = safe_fopen(fname, (char *) "wb", (char *) __FILE__, __LINE__);


    tstart_inner = dtime();
    safe_fwrite(v->tr.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->tr.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->tr.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fwrite(v->tl.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->tl.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->tl.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fwrite(v->br.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->br.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->br.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fwrite(v->bl.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->bl.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fwrite(v->bl.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    /* stop inner timer */
    tend_inner = dtime();

    /* close file and stop outer timer */
    safe_fclose(fname, snapshot, (char *) __FILE__, __LINE__);
    tend_outer = dtime();

    iospeed_inner = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / (tend_inner - tstart_inner);
    iospeed_outer = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / (tend_outer - tstart_outer);

    print_stats("Write snapshot (%lf GB)", TOGB(cellsInVolume * sizeof(real) * 12));
    print_stats("\tInner time %lf seconds (%lf MB/s)", (tend_inner - tstart_inner), iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MB/s)", (tend_outer - tstart_outer), iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);

#endif
};

/*
 * Reads the complete velocity field from disk.
 */
void read_snapshot(char *folder,
                   int suffix,
                   v_t *v,
                   const integer dimmz,
                   const integer dimmx,
                   const integer dimmy) {
#if defined(DO_NOT_PERFORM_IO)
    print_debug("We are not reading the snapshot here cause IO is not enabled!");
#else
    /* local variables */
    double tstart_outer, tstart_inner;
    double iospeed_outer, iospeed_inner;
    double tend_outer, tend_inner;
    const integer cellsInVolume = dimmz * dimmx * dimmy;
    char fname[300];
    int rank = 0;

#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
#endif

    /* open snapshot file and read results */
    sprintf(fname, "%s/snapshot.%03d.%05d", folder, rank, suffix);

    print_debug("[Rank %d] is freading snapshot from %s", rank, fname);

    tstart_outer = dtime();
    FILE *snapshot = safe_fopen(fname, (char *) "rb", (char *) __FILE__, __LINE__);

    tstart_inner = dtime();
    safe_fread(v->tr.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->tr.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->tr.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fread(v->tl.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->tl.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->tl.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fread(v->br.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->br.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->br.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    safe_fread(v->bl.u, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->bl.v, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);
    safe_fread(v->bl.w, sizeof(real), cellsInVolume, snapshot, (char *) __FILE__, __LINE__);

    /* stop inner timer */
    tend_inner = dtime() - tstart_inner;

    /* close file and stop outer timer */
    safe_fclose(fname, snapshot, (char *) __FILE__, __LINE__);
    tend_outer = dtime() - tstart_outer;

    iospeed_inner = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / tend_inner;
    iospeed_outer = ((cellsInVolume * sizeof(real) * 12.f) / (1000.f * 1000.f)) / tend_outer;

    print_stats("Read snapshot (%lf GB)", TOGB(cellsInVolume * sizeof(real) * 12));
    print_stats("\tInner time %lf seconds (%lf MiB/s)", tend_inner, iospeed_inner);
    print_stats("\tOuter time %lf seconds (%lf MiB/s)", tend_outer, iospeed_outer);
    print_stats("\tDifference %lf seconds", tend_outer - tend_inner);
#endif
};

void propagate_shot(time_d direction,
                    v_t v,
                    s_t s,
                    coeff_t coeffs,
                    real *rho,
                    v_t gpu_v,
                    s_t gpu_s,
                    coeff_t gpu_coeffs,
                    real *gpu_rho,
                    int timesteps,
                    int ntbwd,
                    real dt,
                    real dzi,
                    real dxi,
                    real dyi,
                    integer nz0,
                    integer nzf,
                    integer nx0,
                    integer nxf,
                    integer ny0,
                    integer nyf,
                    integer stacki,
                    char *folder,
                    real *UNUSED(dataflush),
                    integer dimmz,
                    integer dimmx,
                    integer dimmy) {
    double tglobal_start, tglobal_total = 0.0;
    double tstress_start, tstress_total = 0.0;
    double tvel_start, tvel_total = 0.0;
    double megacells = 0.0;

    int rank = 0, ranksize = 1;

#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &ranksize);
#endif


    for (int t = 0; t < timesteps; t++) {
        /* print out some information */
        // print_info("[Rank %d]  Computing %d-th timestep", rank, t);

        /* perform IO */
        if (t % stacki == 0 && direction == BACKWARD) read_snapshot(folder, ntbwd - t, &v, dimmz, dimmx, dimmy);

        tglobal_start = dtime();

        /* ------------------------------------------------------------------------------ */
        /*                      VELOCITY COMPUTATION                                      */
        /* ------------------------------------------------------------------------------ */
        /* Phase 1. Computation of the left-most planes of the domain */
        velocity_propagator(v, s, coeffs, rho, gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                            dt, dzi, dxi, dyi,
                            nz0 + HALO,
                            nzf - HALO,
                            nx0 + HALO,
                            nxf - HALO,
                            ny0 + HALO,
                            ny0 + 2 * HALO,
                            dimmz, dimmx);


        /* Phase 1. Computation of the right-most planes of the domain */
        velocity_propagator(v, s, coeffs, rho,  gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                            dt, dzi, dxi, dyi,
                            nz0 + HALO,
                            nzf - HALO,
                            nx0 + HALO,
                            nxf - HALO,
                            nyf - 2 * HALO,
                            nyf - HALO,
                            dimmz, dimmx);

        /* Boundary exchange for velocity values */
        exchange_velocity_boundaries(v, dimmz * dimmx, rank, ranksize, nyf, ny0);

        /* Phase2. Computation of the central planes. */
        tvel_start = dtime();

        velocity_propagator(v, s, coeffs, rho, gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                            dt, dzi, dxi, dyi,
                            nz0 + HALO,
                            nzf - HALO,
                            nx0 + HALO,
                            nxf - HALO,
                            ny0 + HALO,
                            nyf - HALO,
                            dimmz, dimmx);

        tvel_total += (dtime() - tvel_start);


        //debug files
        copy_velocity_data_ToCPU(&v, &s, &coeffs, rho, &gpu_v, &gpu_s, &gpu_coeffs, gpu_rho,dimmx,dimmy,dimmz);

        write_velocity_datafile(&v, &s, &coeffs, rho,dimmx,dimmy,dimmz);

        /* ------------------------------------------------------------------------------ */
        /*                        STRESS COMPUTATION                                      */
        /* ------------------------------------------------------------------------------ */
        /* Phase 1. Computation of the left-most planes of the domain */
        stress_propagator(s, v, coeffs, rho, gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                          dt, dzi, dxi, dyi,
                          nz0 + HALO,
                          nzf - HALO,
                          nx0 + HALO,
                          nxf - HALO,
                          ny0 + HALO,
                          ny0 + 2 * HALO,
                          dimmz, dimmx);

        /* Phase 1. Computation of the right-most planes of the domain */
        stress_propagator(s, v, coeffs, rho, gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                          dt, dzi, dxi, dyi,
                          nz0 + HALO,
                          nzf - HALO,
                          nx0 + HALO,
                          nxf - HALO,
                          nyf - 2 * HALO,
                          nyf - HALO,
                          dimmz, dimmx);

        /* Boundary exchange for stress values */
        exchange_stress_boundaries(s, dimmz * dimmx, rank, ranksize, nyf, ny0);

        /* Phase 2 computation. Central planes of the domain (maingrid) */
        tstress_start = dtime();
        stress_propagator(s, v, coeffs, rho, gpu_v, gpu_s, gpu_coeffs, gpu_rho,
                          dt, dzi, dxi, dyi,
                          nz0 + HALO,
                          nzf - HALO,
                          nx0 + HALO,
                          nxf - HALO,
                          ny0 + HALO,
                          nyf - HALO,
                          dimmz, dimmx);

        tstress_total += (dtime() - tstress_start);
        tglobal_total += (dtime() - tglobal_start);

        /* perform IO */
        if (t % stacki == 0 && direction == FORWARD) write_snapshot(folder, ntbwd - t, &v, dimmz, dimmx, dimmy);
    }

    /* compute some statistics */
    megacells = ((nzf - nz0) * (nxf - nx0) * (nyf - ny0)) / 1e6;
    tglobal_total /= (double) timesteps;
    tstress_total /= (double) timesteps;
    tvel_total /= (double) timesteps;

    print_stats("Maingrid GLOBAL   computation took %lf seconds - %lf Mcells/s", tglobal_total,
                (2 * megacells) / tglobal_total);
    print_stats("Maingrid STRESS   computation took %lf seconds - %lf Mcells/s", tstress_total,
                megacells / tstress_total);
    print_stats("Maingrid VELOCITY computation took %lf seconds - %lf Mcells/s", tvel_total, megacells / tvel_total);
};

/* --------------- BOUNDARY EXCHANGES ---------------------------------------- */
void exchange_velocity_boundaries(v_t v,
                                  const integer plane_size,
                                  const integer rank,
                                  const integer nranks,
                                  const integer nyf,
                                  const integer ny0) {
    const integer num_planes = HALO;
    const integer nelems = num_planes * plane_size;

    const integer left_recv = ny0;
    const integer left_send = ny0 + HALO;

    const integer right_recv = nyf - HALO;
    const integer right_send = nyf - 2 * HALO;

    if (rank != 0) {
        // [RANK-1] <---> [RANK] communication
        EXCHANGE(&v.tl.u[left_send], &v.tl.u[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tl.v[left_send], &v.tl.v[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tl.w[left_send], &v.tl.w[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.tr.u[left_send], &v.tr.u[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tr.v[left_send], &v.tr.v[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tr.w[left_send], &v.tr.w[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.bl.u[left_send], &v.bl.u[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.bl.v[left_send], &v.bl.v[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.bl.w[left_send], &v.bl.w[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.br.u[left_send], &v.br.u[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.br.v[left_send], &v.br.v[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.br.w[left_send], &v.br.w[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
    }

    if (rank != nranks - 1)  //task to exchange stress boundaries
    {
        //                [RANK] <---> [RANK+1] communication
        EXCHANGE(&v.tl.u[right_send], &v.tl.u[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tl.v[right_send], &v.tl.v[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tl.w[right_send], &v.tl.w[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.tr.u[right_send], &v.tr.u[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tr.v[right_send], &v.tr.v[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.tr.w[right_send], &v.tr.w[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.bl.u[right_send], &v.bl.u[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.bl.v[right_send], &v.bl.v[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.bl.w[right_send], &v.bl.w[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&v.br.u[right_send], &v.br.u[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.br.v[right_send], &v.br.v[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&v.br.w[right_send], &v.br.w[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
    }

    print_debug("Velocity boundaries exchanged correctly");
};

void exchange_stress_boundaries(s_t s,
                                const integer plane_size,
                                const integer rank,
                                const integer nranks,
                                const integer nyf,
                                const integer ny0) {
    const integer num_planes = HALO;
    const integer nelems = num_planes * plane_size;

    const integer left_recv = ny0;
    const integer left_send = ny0 + HALO;

    const integer right_recv = nyf - HALO;
    const integer right_send = nyf - 2 * HALO;

    if (rank != 0) {
        // [RANK-1] <---> [RANK] communication
        EXCHANGE(&s.tl.zz[left_send], &s.tl.zz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xz[left_send], &s.tl.xz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.yz[left_send], &s.tl.yz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xx[left_send], &s.tl.xx[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xy[left_send], &s.tl.xy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.yy[left_send], &s.tl.yy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.tr.zz[left_send], &s.tr.zz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xz[left_send], &s.tr.xz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.yz[left_send], &s.tr.yz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xx[left_send], &s.tr.xx[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xy[left_send], &s.tr.xy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.yy[left_send], &s.tr.yy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.bl.zz[left_send], &s.bl.zz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xz[left_send], &s.bl.xz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.yz[left_send], &s.bl.yz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xx[left_send], &s.bl.xx[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xy[left_send], &s.bl.xy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.yy[left_send], &s.bl.yy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.br.zz[left_send], &s.br.zz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xz[left_send], &s.br.xz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.yz[left_send], &s.br.yz[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xx[left_send], &s.br.xx[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xy[left_send], &s.br.xy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.yy[left_send], &s.br.yy[left_recv], rank - 1, rank, nelems, __FILE__, __LINE__);
    }

    if (rank != nranks - 1) {
        //                [RANK] <---> [RANK+1] communication
        EXCHANGE(&s.tl.zz[right_send], &s.tl.zz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xz[right_send], &s.tl.xz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.yz[right_send], &s.tl.yz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xx[right_send], &s.tl.xx[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.xy[right_send], &s.tl.xy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tl.yy[right_send], &s.tl.yy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.tr.zz[right_send], &s.tr.zz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xz[right_send], &s.tr.xz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.yz[right_send], &s.tr.yz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xx[right_send], &s.tr.xx[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.xy[right_send], &s.tr.xy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.tr.yy[right_send], &s.tr.yy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.bl.zz[right_send], &s.bl.zz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xz[right_send], &s.bl.xz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.yz[right_send], &s.bl.yz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xx[right_send], &s.bl.xx[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.xy[right_send], &s.bl.xy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.bl.yy[right_send], &s.bl.yy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);

        EXCHANGE(&s.br.zz[right_send], &s.br.zz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xz[right_send], &s.br.xz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.yz[right_send], &s.br.yz[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xx[right_send], &s.br.xx[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.xy[right_send], &s.br.xy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
        EXCHANGE(&s.br.yy[right_send], &s.br.yy[right_recv], rank + 1, rank, nelems, __FILE__, __LINE__);
    }

    print_debug("Stress boundaries exchanged correctly");
};

void EXCHANGE(const real *sendbuf,
              real *recvbuf,
              const integer dst,
              const integer src,
              const integer message_size,
              const char *file,
              const integer line) {
#if defined(DISTRIBUTED_MEMORY_IMPLEMENTATION)
    int err;
 int tag = 100;

 print_debug( "         [BEFORE]MPI sendrecv [count:%d][dst:%d][src:%d] %s : %d",
           message_size,  dst, src, file, line);

 MPI_Status  statuses[2];
 MPI_Request requests[2];

 MPI_Irecv( recvbuf, message_size, MPI_FLOAT, dst, tag, MPI_COMM_WORLD, &requests[0] );
 MPI_Isend( sendbuf, message_size, MPI_FLOAT, dst, tag, MPI_COMM_WORLD, &requests[1] );
 err = MPI_Waitall(2, requests, statuses);

 print_debug( "         [AFTER ]MPI sendrecv                          %s : %d",
           file, line);

 if ( err != MPI_SUCCESS )
   {
       print_error("MPI error %d!", err);
       abort();
   }

#endif
};

